#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include <>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
glm::vec3* dev_rearrangePos;
glm::vec3* dev_rearrangeVel;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.

  // Part-2.2
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");
  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");
  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");
  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);
  dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);

  // Part-2.3
  hipMalloc((void**)&dev_rearrangePos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_rearrangePos failed!");

  hipMalloc((void**)&dev_rearrangeVel, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_rearrangeVel failed!");

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
  // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
  // Rule 2: boids try to stay a distance d away from each other
  // Rule 3: boids try to match the speed of surrounding boids
  

  // Rule 1
  int rule1NumOfNeighbor = 0;
  glm::vec3 perceived_center(0, 0, 0);
  // Rule 2
  glm::vec3 rule2Vel(0, 0, 0);        
  // Rule 3
  glm::vec3 rule3Vel(0, 0, 0);
  int rule3NumOfNeighbor = 0;

  for (int i = 0; i < N; ++i)
  {
    if (i == iSelf)
      continue;

    if (glm::distance(pos[i], pos[iSelf]) < rule1Distance)
    {
      ++rule1NumOfNeighbor;
      perceived_center += pos[i];
    }

    if (glm::distance(pos[i], pos[iSelf]) < rule2Distance)
    {
      rule2Vel -= (pos[i] - pos[iSelf]);
    }

    if (glm::distance(pos[i], pos[iSelf]) < rule3Distance)
    {
      ++rule3NumOfNeighbor;
      rule3Vel += vel[i];
    }
  }


  glm::vec3 ret(0, 0, 0);

  if (rule1NumOfNeighbor > 0)
  {
    perceived_center /= rule1NumOfNeighbor;
    ret += (perceived_center - pos[iSelf]) * rule1Scale;
  }

  ret += rule2Vel * rule2Scale;

  if (rule3NumOfNeighbor > 0)
  {
    rule3Vel /= rule3NumOfNeighbor;
    ret += rule3Vel  * rule3Scale;
  }

  return ret;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  // Compute a new velocity based on pos and vel1
  // Clamp the speed
  // Record the new velocity into vel2. Question: why NOT vel1?
  
  int index = (blockDim.x * blockIdx.x) + threadIdx.x;

  if (index >= N)
    return;

  glm::vec3 newVel = vel1[index] + computeVelocityChange(N, index, pos, vel1);
  
  newVel = glm::clamp(newVel, -1 * maxSpeed, 1 * maxSpeed);

  vel2[index] = newVel;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2

  int index = (blockDim.x * blockIdx.x) + threadIdx.x;

  if (index >= N)
    return;

  glm::vec3 boidPos = pos[index];

  int x = glm::floor((boidPos[0] - gridMin[0]) * inverseCellWidth);
  int y = glm::floor((boidPos[1] - gridMin[1]) * inverseCellWidth);
  int z = glm::floor((boidPos[2] - gridMin[2]) * inverseCellWidth);
  int gridIndex = gridIndex3Dto1D(x, y, z, gridResolution);

  gridIndices[index] = gridIndex;
  indices[index] = index; // Array indices helps tracking the pos and dev_velX after sorting 
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"

  int index = (blockDim.x * blockIdx.x) + threadIdx.x;

  if (index >= N)
    return;

  if (index == 0)
  {
    gridCellStartIndices[particleGridIndices[index]] = index;
  }
  else if(index == N - 1)
  {
    gridCellEndIndices[particleGridIndices[index]] = index;
  }
  else
  {
    if (particleGridIndices[index - 1] != particleGridIndices[index])
    {
      gridCellEndIndices[particleGridIndices[index - 1]] = index - 1;
      gridCellStartIndices[particleGridIndices[index]] = index;
    }
  }
}

__device__ void computeVelocityChangeSearchScattered(int gridIndex, int boidIndex, glm::vec3 boidPos,
  int& rule1Num, int& rule3Num, glm::vec3& rule1Center, glm::vec3& rule2Vel, glm::vec3& rule3Vel,
  int* gridCellStartIndices, int* gridCellEndIndices, int* particleArrayIndices, glm::vec3* pos, glm::vec3* vel)
{
  // If no particles inside this grid
  //if (gridCellStartIndices[gridIndex] < 0)
  //  return;

  int iNeighbor;
  float distance;
  glm::vec3 otherPos;
  for (int i = gridCellStartIndices[gridIndex]; i >= 0 && i <= gridCellEndIndices[gridIndex]; ++i)
  {
    iNeighbor = particleArrayIndices[i];

    if (iNeighbor == boidIndex)
      continue;

    otherPos = pos[iNeighbor];

    distance = glm::distance(boidPos, otherPos);

    // Rule1
    if (distance < rule1Distance)
    {
      rule1Num = rule1Num + 1;
      rule1Center += otherPos;
    }

    // Rule2
    if (distance < rule2Distance)
    {
      rule2Vel -= (otherPos - boidPos);
    }

    // Rule3
    if (distance < rule3Distance)
    {
      rule3Num = rule3Num + 1;
      rule3Vel += vel[iNeighbor];
    }
  }
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2

  int index = (blockDim.x * blockIdx.x) + threadIdx.x;

  if (index >= N)
    return;

  glm::vec3 boidPos = pos[index];

  float xGrid = (boidPos[0] - gridMin[0]) * inverseCellWidth;
  float yGrid = (boidPos[1] - gridMin[1]) * inverseCellWidth;
  float zGrid = (boidPos[2] - gridMin[2]) * inverseCellWidth;

  // All neighbors including current gird
  int x = glm::floor(xGrid);
  int y = glm::floor(yGrid);
  int z = glm::floor(zGrid);
  int x2 = glm::fract(xGrid) > 0.5 ? x + 1 : x - 1;
  int y2 = glm::fract(yGrid) > 0.5 ? y + 1 : y - 1;
  int z2 = glm::fract(zGrid) > 0.5 ? z + 1 : z - 1;

  int rule1Num = 0;
  int rule3Num = 0;
  glm::vec3 rule1Center(0, 0, 0);
  glm::vec3 rule2Vel(0, 0, 0);
  glm::vec3 rule3Vel(0, 0, 0);

  // Traverse the 8 neighbor grids
  int gridIndex1 = gridIndex3Dto1D(x, y, z, gridResolution);
  int gridIndex2 = gridIndex3Dto1D(x2, y, z, gridResolution);
  int gridIndex3 = gridIndex3Dto1D(x, y2, z, gridResolution);
  int gridIndex4 = gridIndex3Dto1D(x, y, z2, gridResolution);
  int gridIndex5 = gridIndex3Dto1D(x2, y2, z, gridResolution);
  int gridIndex6 = gridIndex3Dto1D(x2, y, z2, gridResolution);
  int gridIndex7 = gridIndex3Dto1D(x, y2, z2, gridResolution);
  int gridIndex8 = gridIndex3Dto1D(x2, y2, z2, gridResolution);

  computeVelocityChangeSearchScattered(gridIndex1, index, boidPos, rule1Num, rule3Num, rule1Center, rule2Vel, rule3Vel,
    gridCellStartIndices, gridCellEndIndices, particleArrayIndices, pos, vel1);
  computeVelocityChangeSearchScattered(gridIndex2, index, boidPos, rule1Num, rule3Num, rule1Center, rule2Vel, rule3Vel,
    gridCellStartIndices, gridCellEndIndices, particleArrayIndices, pos, vel1);
  computeVelocityChangeSearchScattered(gridIndex3, index, boidPos, rule1Num, rule3Num, rule1Center, rule2Vel, rule3Vel,
    gridCellStartIndices, gridCellEndIndices, particleArrayIndices, pos, vel1);
  computeVelocityChangeSearchScattered(gridIndex4, index, boidPos, rule1Num, rule3Num, rule1Center, rule2Vel, rule3Vel,
    gridCellStartIndices, gridCellEndIndices, particleArrayIndices, pos, vel1);
  computeVelocityChangeSearchScattered(gridIndex5, index, boidPos, rule1Num, rule3Num, rule1Center, rule2Vel, rule3Vel,
    gridCellStartIndices, gridCellEndIndices, particleArrayIndices, pos, vel1);
  computeVelocityChangeSearchScattered(gridIndex6, index, boidPos, rule1Num, rule3Num, rule1Center, rule2Vel, rule3Vel,
    gridCellStartIndices, gridCellEndIndices, particleArrayIndices, pos, vel1);
  computeVelocityChangeSearchScattered(gridIndex7, index, boidPos, rule1Num, rule3Num, rule1Center, rule2Vel, rule3Vel,
    gridCellStartIndices, gridCellEndIndices, particleArrayIndices, pos, vel1);
  computeVelocityChangeSearchScattered(gridIndex8, index, boidPos, rule1Num, rule3Num, rule1Center, rule2Vel, rule3Vel,
    gridCellStartIndices, gridCellEndIndices, particleArrayIndices, pos, vel1);

  // For detect 3x3x3 grids
  //for (int i = x - 1; i >= 0 && i <= x + 1; ++i)
  //{
  //  for (int j = y - 1; j >= 0 && j <= y + 1; ++j)
  //  {
  //    for (int k = z - 1; k >= 0 && k <= z + 1; ++k)
  //    {
  //      computeVelocityChangeSearchScattered(gridIndex3Dto1D(i, j, k, gridResolution), index, boidPos, rule1Num, rule3Num, rule1Center, rule2Vel, rule3Vel,
  //        gridCellStartIndices, gridCellEndIndices, particleArrayIndices, pos, vel1);
  //    }
  //  }
  //}

  glm::vec3 ret(0, 0, 0);

  if (rule1Num > 0)
  {
    rule1Center /= rule1Num;
    ret += (rule1Center - boidPos) * rule1Scale;
  }

  ret += rule2Vel * rule2Scale;

  if (rule3Num > 0)
  {
    rule3Vel /= rule3Num;
    ret += rule3Vel * rule3Scale;
  }

  float retX = ret[0];
  float retY = ret[1];
  float retZ = ret[2];

  vel2[index] = glm::clamp(vel1[index] + ret, -1 * maxSpeed, maxSpeed);
}

__device__ void computeVelocityChangeSearchCoherent(int gridIndex, int boidIndex, glm::vec3 boidPos,
  int& rule1Num, int& rule3Num, glm::vec3& rule1Center, glm::vec3& rule2Vel, glm::vec3& rule3Vel,
  int* gridCellStartIndices, int* gridCellEndIndices, glm::vec3* rearrangePos, glm::vec3* reaarangeVel)
{
  // If no particles inside this grid
  //if (gridCellStartIndices[gridIndex] < 0)
  //  return;

  float distance;
  glm::vec3 otherPos;
  for (int i = gridCellStartIndices[gridIndex]; i >= 0 && i <= gridCellEndIndices[gridIndex]; ++i)
  {
    if (i == boidIndex)
      continue;

    otherPos = rearrangePos[i];

    distance = glm::distance(boidPos, otherPos);

    // Rule1
    if (distance < rule1Distance)
    {
      rule1Num = rule1Num + 1;
      rule1Center += otherPos;
    }

    // Rule2
    if (distance < rule2Distance)
    {
      rule2Vel -= (otherPos - boidPos);
    }

    // Rule3
    if (distance < rule3Distance)
    {
      rule3Num = rule3Num + 1;
      rule3Vel += reaarangeVel[i];
    }
  }
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2

  int index = (blockDim.x * blockIdx.x) + threadIdx.x;

  if (index >= N)
    return;

  glm::vec3 boidPos = pos[index];

  float xGrid = (boidPos[0] - gridMin[0]) * inverseCellWidth;
  float yGrid = (boidPos[1] - gridMin[1]) * inverseCellWidth;
  float zGrid = (boidPos[2] - gridMin[2]) * inverseCellWidth;

  // All neighbors including current gird
  int x = glm::floor(xGrid);
  int y = glm::floor(yGrid);
  int z = glm::floor(zGrid);
  int x2 = glm::fract(xGrid) > 0.5 ? x + 1 : x - 1;
  int y2 = glm::fract(yGrid) > 0.5 ? y + 1 : y - 1;
  int z2 = glm::fract(zGrid) > 0.5 ? z + 1 : z - 1;

  int rule1Num = 0;
  int rule3Num = 0;
  glm::vec3 rule1Center(0, 0, 0);
  glm::vec3 rule2Vel(0, 0, 0);
  glm::vec3 rule3Vel(0, 0, 0);

  // Traverse the 8 neighbor grids
  int gridIndex1 = gridIndex3Dto1D(x, y, z, gridResolution);
  int gridIndex2 = gridIndex3Dto1D(x2, y, z, gridResolution);
  int gridIndex3 = gridIndex3Dto1D(x, y2, z, gridResolution);
  int gridIndex4 = gridIndex3Dto1D(x, y, z2, gridResolution);
  int gridIndex5 = gridIndex3Dto1D(x2, y2, z, gridResolution);
  int gridIndex6 = gridIndex3Dto1D(x2, y, z2, gridResolution);
  int gridIndex7 = gridIndex3Dto1D(x, y2, z2, gridResolution);
  int gridIndex8 = gridIndex3Dto1D(x2, y2, z2, gridResolution);

  computeVelocityChangeSearchCoherent(gridIndex1, index, boidPos, rule1Num, rule3Num, rule1Center, rule2Vel, rule3Vel,
    gridCellStartIndices, gridCellEndIndices, pos, vel1);
  computeVelocityChangeSearchCoherent(gridIndex2, index, boidPos, rule1Num, rule3Num, rule1Center, rule2Vel, rule3Vel,
    gridCellStartIndices, gridCellEndIndices, pos, vel1);
  computeVelocityChangeSearchCoherent(gridIndex3, index, boidPos, rule1Num, rule3Num, rule1Center, rule2Vel, rule3Vel,
    gridCellStartIndices, gridCellEndIndices, pos, vel1);
  computeVelocityChangeSearchCoherent(gridIndex4, index, boidPos, rule1Num, rule3Num, rule1Center, rule2Vel, rule3Vel,
    gridCellStartIndices, gridCellEndIndices, pos, vel1);
  computeVelocityChangeSearchCoherent(gridIndex5, index, boidPos, rule1Num, rule3Num, rule1Center, rule2Vel, rule3Vel,
    gridCellStartIndices, gridCellEndIndices, pos, vel1);
  computeVelocityChangeSearchCoherent(gridIndex6, index, boidPos, rule1Num, rule3Num, rule1Center, rule2Vel, rule3Vel,
    gridCellStartIndices, gridCellEndIndices, pos, vel1);
  computeVelocityChangeSearchCoherent(gridIndex7, index, boidPos, rule1Num, rule3Num, rule1Center, rule2Vel, rule3Vel,
    gridCellStartIndices, gridCellEndIndices, pos, vel1);
  computeVelocityChangeSearchCoherent(gridIndex8, index, boidPos, rule1Num, rule3Num, rule1Center, rule2Vel, rule3Vel,
    gridCellStartIndices, gridCellEndIndices, pos, vel1);

  // For detect 3x3x3 grids
  //for (int i = x - 1; i >= 0 && i <= x + 1; ++i)
  //{
  //  for (int j = y - 1; j >= 0 && j <= y + 1; ++j)
  //  {
  //    for (int k = z - 1; k >= 0 && k <= z + 1; ++k)
  //    {
  //      computeVelocityChangeSearchCoherent(gridIndex3Dto1D(i, j, k, gridResolution), index, boidPos, rule1Num, rule3Num, rule1Center, rule2Vel, rule3Vel,
  //        gridCellStartIndices, gridCellEndIndices, pos, vel1);
  //    }
  //  }
  //}

  glm::vec3 ret(0, 0, 0);

  if (rule1Num > 0)
  {
    rule1Center /= rule1Num;
    ret += (rule1Center - boidPos) * rule1Scale;
  }

  ret += rule2Vel * rule2Scale;

  if (rule3Num > 0)
  {
    rule3Vel /= rule3Num;
    ret += rule3Vel * rule3Scale;
  }

  float retX = ret[0];
  float retY = ret[1];
  float retZ = ret[2];

  vel2[index] = glm::clamp(vel1[index] + ret, -1 * maxSpeed, maxSpeed);
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
  // TODO-1.2 ping-pong the velocity buffers

  int N = numObjects;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  kernUpdateVelocityBruteForce<<<fullBlocksPerGrid,blockSize>>>(N, dev_pos, dev_vel1, dev_vel2);
  kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(N, dt, dev_pos, dev_vel2);
  hipMemcpy(dev_vel1, dev_vel2, N * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed


  int N = numObjects;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // Setup indices and grid indices
  kernComputeIndices<<<fullBlocksPerGrid, blockSize>>>(N, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

  // Sort
  thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects,
    dev_thrust_particleArrayIndices);
 
  // Setup start and end indices
  dim3 fullBlocksPerGridCell((gridCellCount + blockSize - 1) / blockSize);
  kernResetIntBuffer<<<fullBlocksPerGridCell, blockSize>>>(gridCellCount, dev_gridCellStartIndices, -1);
  kernResetIntBuffer<<<fullBlocksPerGridCell, blockSize>>>(gridCellCount, dev_gridCellEndIndices, -1);
  kernIdentifyCellStartEnd<<<fullBlocksPerGrid, blockSize>>>(N, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

  // Calculate velocity
  kernUpdateVelNeighborSearchScattered<<<fullBlocksPerGrid, blockSize>>>(N, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices,
    dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);

  // Update position
  kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(N, dt, dev_pos, dev_vel2);

  // Swap buffer
  hipMemcpy(dev_vel1, dev_vel2, N * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
}

__global__ void computeRearrangeArray(int N, int* particleArrayIndices, glm::vec3* originArray, glm::vec3* rearrangeArray)
{
  int index = (blockDim.x * blockIdx.x) + threadIdx.x;

  if (index >= N)
    return;

  int newIndex = particleArrayIndices[index];
  rearrangeArray[index] = originArray[newIndex];
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.

  int N = numObjects;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // Setup indices and grid indices
  kernComputeIndices <<<fullBlocksPerGrid, blockSize >>> (N, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

  // Sort
  thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects,
    dev_thrust_particleArrayIndices);

  // Setup start and end indices
  dim3 fullBlocksPerGridCell((gridCellCount + blockSize - 1) / blockSize);
  kernResetIntBuffer <<<fullBlocksPerGridCell, blockSize >>> (gridCellCount, dev_gridCellStartIndices, -1);
  kernResetIntBuffer <<<fullBlocksPerGridCell, blockSize >>> (gridCellCount, dev_gridCellEndIndices, -1);
  kernIdentifyCellStartEnd <<<fullBlocksPerGrid, blockSize >>> (N, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

  // Rearrange pos and vel
  computeRearrangeArray <<<fullBlocksPerGrid, blockSize >>> (N, dev_particleArrayIndices, dev_pos, dev_rearrangePos);
  computeRearrangeArray <<<fullBlocksPerGrid, blockSize >>> (N, dev_particleArrayIndices, dev_vel1, dev_rearrangeVel);

  // Calulate velocity
  kernUpdateVelNeighborSearchCoherent <<<fullBlocksPerGrid, blockSize >>> (N, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices,
    dev_rearrangePos, dev_rearrangeVel, dev_vel2);

  // Update position
  kernUpdatePos <<<fullBlocksPerGrid, blockSize >>> (N, dt, dev_rearrangePos, dev_vel2);
  hipMemcpy(dev_pos, dev_rearrangePos, N * sizeof(glm::vec3), hipMemcpyDeviceToDevice);

  // Swap buffer
  hipMemcpy(dev_vel1, dev_vel2, N * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);

  hipFree(dev_rearrangePos);
  hipFree(dev_rearrangeVel);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
