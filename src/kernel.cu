#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"
#include <>

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

#define USE_DOUBLE_CELL_WIDTH 1

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
glm::vec3 *dev_reshuffledPos;
glm::vec3 *dev_reshuffledVel;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
#if USE_DOUBLE_CELL_WIDTH
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
#else
  gridCellWidth = std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
#endif
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));


  dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);
  dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);

  hipMalloc((void**)&dev_reshuffledPos, N * sizeof(glm::vec3));
  hipMalloc((void**)&dev_reshuffledVel, N * sizeof(glm::vec3));

  hipDeviceSynchronize();

}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
  glm::vec3 velChange;
  // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
  glm::vec3 perceivedCenter;
  int numNeighbors = 0;
  for (int i = 0; i < N; ++i)
  {
    if (i != iSelf && glm::distance(pos[i], pos[iSelf]) < rule1Distance)
    {
      perceivedCenter += pos[i];
      numNeighbors++;
    }
  }
  if (numNeighbors > 0)
  {
    perceivedCenter /= numNeighbors;
    velChange += (perceivedCenter - pos[iSelf]) * rule1Scale;
  }

  // Rule 2: boids try to stay a distance d away from each other
  glm::vec3 c;
  for (int i = 0; i < N; ++i)
  {
    if (i != iSelf && glm::distance(pos[i], pos[iSelf]) < rule2Distance)
    {
      c -= (pos[i] - pos[iSelf]);
    }
  }
  velChange += c * rule2Scale;

  // Rule 3: boids try to match the speed of surrounding boids
  glm::vec3 perceivedVel;
  numNeighbors = 0;
  for (int i = 0; i < N; ++i)
  {
    if (i != iSelf && glm::distance(pos[i], pos[iSelf]) < rule3Distance)
    {
      perceivedVel += vel[i];
      numNeighbors++;
    }
  }
  if (numNeighbors > 0)
  {
    perceivedVel /= numNeighbors;
    velChange += perceivedVel * rule3Scale;
  }

  return velChange;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  // Compute a new velocity based on pos and vel1
  // Clamp the speed
  // Record the new velocity into vel2. Question: why NOT vel1?

  int currIndex = threadIdx.x + (blockIdx.x * blockDim.x);
  glm::vec3 newVel = computeVelocityChange(N, currIndex, pos, vel1) + vel1[currIndex];
  float newSpeed = glm::length(newVel);
  if (newSpeed > maxSpeed)
  {
    newVel = newVel / newSpeed * maxSpeed;
  }
  vel2[currIndex] = newVel;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index >= N) return;
  
  indices[index] = index;
  
  glm::vec3 gridIndex3D = glm::floor((pos[index] - gridMin) * inverseCellWidth);
  gridIndices[index] = gridIndex3Dto1D(gridIndex3D.x, gridIndex3D.y, gridIndex3D.z, gridResolution);
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"

  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index >= N) return;
  
  int currGridIndex = particleGridIndices[index];
  
  if (index > 0)
  {
    int lastGridIndex = particleGridIndices[index - 1];
    if (lastGridIndex != currGridIndex)
    {
      gridCellStartIndices[currGridIndex] = index;
      gridCellEndIndices[lastGridIndex] = index - 1;
    }
  }
  // First one must be the start
  else
  {
    gridCellStartIndices[currGridIndex] = index;
  }
  // Last one must be the end
  if (index == N - 1)
  {
    gridCellEndIndices[currGridIndex] = index;
  }  
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2

  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index >= N) return;

  glm::vec3 gridIndex3D = (pos[index] - gridMin) * inverseCellWidth;

  // relativeNeighborGrid is the direction (-1 or 1) in which grid may contain neighbors of the boid
  glm::ivec3 relativeNeighborGrid = glm::round(glm::fract(gridIndex3D)) * 2.f - 1.f;
  glm::ivec3 zeroVec3;
  glm::ivec3 directions3D[2] = { zeroVec3, relativeNeighborGrid };
  gridIndex3D = glm::floor(gridIndex3D);

  glm::vec3 velChange;
  // variables for rule 1
  glm::vec3 perceivedCenter;
  int numNeighbors1 = 0;
  // variables for rule 2
  glm::vec3 c;
  // varirables for rule 3
  glm::vec3 perceivedVel;
  int numNeighbors3 = 0;
  // Find out the cells which may contain neighbors
  for (int i = 0; i < 2; ++i)
  {
    glm::ivec3 neighborGridIndex3D;
    neighborGridIndex3D.x = gridIndex3D.x + directions3D[i].x;
    // out of bound check
    if (neighborGridIndex3D.x < 0 || neighborGridIndex3D.x >= gridResolution) continue;
    for (int j = 0; j < 2; ++j)
    {
      neighborGridIndex3D.y = gridIndex3D.y + directions3D[j].y;
      if (neighborGridIndex3D.y < 0 || neighborGridIndex3D.y >= gridResolution) continue;
      for (int k = 0; k < 2; ++k)
      {
        neighborGridIndex3D.z = gridIndex3D.z + directions3D[k].z;
        if (neighborGridIndex3D.z < 0 || neighborGridIndex3D.z >= gridResolution) continue;

        int gridCell = gridIndex3Dto1D(neighborGridIndex3D.x, neighborGridIndex3D.y, neighborGridIndex3D.z, gridResolution);

        if (gridCellStartIndices[gridCell] == -1) continue;
        // Check all of boids in this cell
        for (int neighborArrayIndex = gridCellStartIndices[gridCell]; neighborArrayIndex <= gridCellEndIndices[gridCell]; ++neighborArrayIndex)
        {
          int iNeighbor = particleArrayIndices[neighborArrayIndex];
          if (iNeighbor == index) continue;
          float dist = glm::distance(pos[iNeighbor], pos[index]);
          // Rule 1
          if (dist < rule1Distance)
          {
            perceivedCenter += pos[iNeighbor];
            numNeighbors1++;
          }
          //Rule 2
          if (dist < rule2Distance)
          {
            c -= (pos[iNeighbor] - pos[index]);
          }
          //Rule 3
          if (dist < rule3Distance)
          {
            perceivedVel += vel1[iNeighbor];
            numNeighbors3++;
          }
        }
      }
    }
  }

  // Rule 1
  if (numNeighbors1 > 0)
  {
    perceivedCenter /= numNeighbors1;
    velChange += (perceivedCenter - pos[index]) * rule1Scale;
  }
  // Rule 2
  velChange += c * rule2Scale;
  // Rule 3
  if (numNeighbors3 > 0)
  {
    perceivedVel /= numNeighbors3;
    velChange += perceivedVel * rule3Scale;
  }

  // compute new velocity
  glm::vec3 newVel = velChange + vel1[index];
  float newSpeed = glm::length(newVel);
  if (newSpeed > maxSpeed)
  {
    newVel = newVel / newSpeed * maxSpeed;
  }
  vel2[index] = newVel;
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index >= N) return;

  glm::vec3 gridIndex3D = (pos[index] - gridMin) * inverseCellWidth;

  // relativeNeighborGrid is the direction (-1 or 1) in which grid may contain neighbors of the boid
  glm::ivec3 relativeNeighborGrid = glm::round(glm::fract(gridIndex3D)) * 2.f - 1.f;
  glm::ivec3 zeroVec3;
  glm::ivec3 directions3D[2] = { zeroVec3, relativeNeighborGrid };
  gridIndex3D = glm::floor(gridIndex3D);

  glm::vec3 velChange;
  // variables for rule 1
  glm::vec3 perceivedCenter;
  int numNeighbors1 = 0;
  // variables for rule 2
  glm::vec3 c;
  // varirables for rule 3
  glm::vec3 perceivedVel;
  int numNeighbors3 = 0;
  // Find out the cells which may contain neighbors
#if USE_DOUBLE_CELL_WIDTH
  for (int i = 0; i < 2; ++i)
  {
    glm::ivec3 neighborGridIndex3D;
    neighborGridIndex3D.x = gridIndex3D.x + directions3D[i].x;
    // out of bound check
    if (neighborGridIndex3D.x < 0 || neighborGridIndex3D.x >= gridResolution) continue;
    for (int j = 0; j < 2; ++j)
    {
      neighborGridIndex3D.y = gridIndex3D.y + directions3D[j].y;
      if (neighborGridIndex3D.y < 0 || neighborGridIndex3D.y >= gridResolution) continue;
      for (int k = 0; k < 2; ++k)
      {
        neighborGridIndex3D.z = gridIndex3D.z + directions3D[k].z;
        if (neighborGridIndex3D.z < 0 || neighborGridIndex3D.z >= gridResolution) continue;
#else
  for (int i = -1; i <= 1; ++i)
  {
    glm::ivec3 neighborGridIndex3D;
    neighborGridIndex3D.x = gridIndex3D.x + i;
    // out of bound check
    if (neighborGridIndex3D.x < 0 || neighborGridIndex3D.x >= gridResolution) continue;
    for (int j = -1; j <= 1; ++j)
    {
      neighborGridIndex3D.y = gridIndex3D.y + j;
      if (neighborGridIndex3D.y < 0 || neighborGridIndex3D.y >= gridResolution) continue;
      for (int k = -1; k <= 1; ++k)
      {
        neighborGridIndex3D.z = gridIndex3D.z + k;
        if (neighborGridIndex3D.z < 0 || neighborGridIndex3D.z >= gridResolution) continue;
#endif

        int gridCell = gridIndex3Dto1D(neighborGridIndex3D.x, neighborGridIndex3D.y, neighborGridIndex3D.z, gridResolution);

        if (gridCellStartIndices[gridCell] == -1) continue;
        // Check all of boids in this cell
        for (int iNeighbor = gridCellStartIndices[gridCell]; iNeighbor <= gridCellEndIndices[gridCell]; ++iNeighbor)
        {
          if (iNeighbor == index) continue;
          float dist = glm::distance(pos[iNeighbor], pos[index]);
          // Rule 1
          if (dist < rule1Distance)
          {
            perceivedCenter += pos[iNeighbor];
            numNeighbors1++;
          }
          //Rule 2
          if (dist < rule2Distance)
          {
            c -= (pos[iNeighbor] - pos[index]);
          }
          //Rule 3
          if (dist < rule3Distance)
          {
            perceivedVel += vel1[iNeighbor];
            numNeighbors3++;
          }
        }
      }
    }
  }

  // Rule 1
  if (numNeighbors1 > 0)
  {
    perceivedCenter /= numNeighbors1;
    velChange += (perceivedCenter - pos[index]) * rule1Scale;
  }
  // Rule 2
  velChange += c * rule2Scale;
  // Rule 3
  if (numNeighbors3 > 0)
  {
    perceivedVel /= numNeighbors3;
    velChange += perceivedVel * rule3Scale;
  }

  // compute new velocity
  glm::vec3 newVel = velChange + vel1[index];
  float newSpeed = glm::length(newVel);
  if (newSpeed > maxSpeed)
  {
    newVel = newVel / newSpeed * maxSpeed;
  }
  vel2[index] = newVel;

}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
  // TODO-1.2 ping-pong the velocity buffers
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
  kernUpdateVelocityBruteForce<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, dev_vel1, dev_vel2);
  kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);
  std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernComputeIndices<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount, gridMinimum, 
    gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
  
  thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, 
    dev_thrust_particleArrayIndices);

  dim3 fullBlocksPerGridCell((gridCellCount + blockSize - 1) / blockSize);
  kernResetIntBuffer<<<fullBlocksPerGridCell, blockSize>>>(gridCellCount, dev_gridCellStartIndices, -1);
  kernResetIntBuffer<<<fullBlocksPerGridCell, blockSize>>>(gridCellCount, dev_gridCellEndIndices, -1);
  kernIdentifyCellStartEnd<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_particleGridIndices, 
    dev_gridCellStartIndices, dev_gridCellEndIndices);

  kernUpdateVelNeighborSearchScattered<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth,
    gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);

  kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);
  std::swap(dev_vel1, dev_vel2);

}

__global__ void kernReshuffleVec3Buffer(int N, int* oldIndices, glm::vec3* oldBuffer, glm::vec3* newBuffer)
{
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index >= N) return;

  int oldIndex = oldIndices[index];
  newBuffer[index] = oldBuffer[oldIndex];
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernComputeIndices<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount, gridMinimum,
    gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

  thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects,
    dev_thrust_particleArrayIndices);

  dim3 fullBlocksPerGridCell((gridCellCount + blockSize - 1) / blockSize);
  kernResetIntBuffer<<<fullBlocksPerGridCell, blockSize>>>(gridCellCount, dev_gridCellStartIndices, -1);
  kernResetIntBuffer<<<fullBlocksPerGridCell, blockSize>>>(gridCellCount, dev_gridCellEndIndices, -1);
  kernIdentifyCellStartEnd<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_particleGridIndices,
    dev_gridCellStartIndices, dev_gridCellEndIndices);

  //Use the rearranged array index buffer to reshuffle all the particle data in the simulation array
  kernReshuffleVec3Buffer<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_particleArrayIndices, dev_pos, dev_reshuffledPos);
  kernReshuffleVec3Buffer<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_particleArrayIndices, dev_vel1, dev_reshuffledVel);

  kernUpdateVelNeighborSearchCoherent<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth,
    gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_reshuffledPos, dev_reshuffledVel, dev_vel2);

  kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_reshuffledPos, dev_vel2);
  std::swap(dev_vel1, dev_vel2);
  std::swap(dev_pos, dev_reshuffledPos);

}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);

  hipFree(dev_reshuffledPos);
  hipFree(dev_reshuffledVel);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
