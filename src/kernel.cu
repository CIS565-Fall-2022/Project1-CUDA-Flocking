#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"
#include <>

// LOOK-2.1 potentially useful for doing grid-based neighbor search
// Di: Returns max of the two or min of the two elements for comparison.
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f // 5.0f
#define rule2Distance 3.0f // 3.0f
#define rule3Distance 5.0f // 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
// Di all initialized on GPU
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?

// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
// glm::vec3* dev_posSortedPrev;
glm::vec3* dev_posSorted;
glm::vec3* dev_vel1Sorted;
// thrust::device_ptr<int> dev_thrust_particleArrayIndicesValue;


// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  // some threads are extra and thus won't enter this case.
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // di added
  hipMalloc((void**)&dev_posSorted, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_posSorted failed!");

  hipMalloc((void**)&dev_vel1Sorted, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_posSortedVel1 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);
  dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);

  // dev_posSortedPrev = dev_pos;

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO <<<fullBlocksPerGrid, blockSize >>>(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO <<<fullBlocksPerGrid, blockSize >>>(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
  // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
  // Rule 2: boids try to stay a distance d away from each other
  // Rule 3: boids try to match the speed of surrounding boids
  //// return glm::vec3(0.0f, 0.0f, 0.0f);
    glm::vec3 velocityChange = glm::vec3(0.f);
    glm::vec3 selfPos = pos[iSelf];
 
    // Rule 1
    // find the average position of all the birds
    glm::vec3 center = glm::vec3(0.f);
    int numValid1 = 0;

    glm::vec3 c = glm::vec3(0.f);

    glm::vec3 perceivedVelocity = glm::vec3(0.f);
    int numValid3 = 0;

    for (int i = 0; i < N; i++) {
        glm::vec3 bPos = pos[i];
        float dist = glm::distance(selfPos, bPos);

        // Rule 1
        if (i != iSelf && dist < rule1Distance) {
            center += pos[i];
            numValid1 += 1;
        }

        // Rule 2
        if (i != iSelf && dist < rule2Distance) {
            c -= (bPos - selfPos);
        }

        // Rule 3
        if (i != iSelf && dist < rule3Distance) {
            perceivedVelocity += vel[i];
            numValid3 += 1;
        }

    }

    if (numValid1 != 0) {
        center /= numValid1;
        velocityChange += (center - selfPos) * rule1Scale;
    }

    velocityChange += c * rule2Scale;

    if (numValid3 != 0) {
        perceivedVelocity /= numValid3;
        velocityChange += perceivedVelocity * rule3Scale;
    }

    return velocityChange;

}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  // Compute a new velocity based on pos and vel1
  // Clamp the speed
  // Record the new velocity into vel2. Question: why NOT vel1? -> 
 
    // we don't want to overwrite original velocities because other boids still need to refer to it to compute their vels.
    // and vel1 and vel2 are all being accessed at the same time so if info in vel1 gets changed while another thread is accessing
    // that would be bad.
    // once we finish computing these vels, vel1 becomes our "write" array and vel2 becomes our "read" array
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) {
        return;
    }
    glm::vec3 velChange = computeVelocityChange(N, index, pos, vel1);
    glm::vec3 finalVel = vel1[index] + velChange;
    float speed = glm::length(finalVel);
    if (speed > maxSpeed) {
        finalVel = finalVel * maxSpeed / speed;
    }
    vel2[index] = finalVel;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];

  // debug value
  glm::vec3 thisVel = vel[index];

  thisPos += vel[index] * dt;

  // printf("vel x: %f, vel y: %f, vel z: %f", vel[index].x, vel[index].y, vel[index].z);

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridSideCount) {
  return x + y * gridSideCount + z * gridSideCount * gridSideCount;
}

__global__ void kernComputeIndices(int N, int gridSideCount,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2
    
    // this kern function is called in parallel by each boid so it only needs to be 
    int boidIdx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (boidIdx < N) {
        // compute components
        glm::vec3 boidPos = pos[boidIdx];

        int ix = (int) (std::floor((boidPos.x - gridMin.x) * inverseCellWidth));
        int iy = (int) (std::floor((boidPos.y - gridMin.y) * inverseCellWidth));
        int iz = (int) (std::floor((boidPos.z - gridMin.z) * inverseCellWidth));

        // compute 1D cell idx from 3D
        int cellIdx = gridIndex3Dto1D(ix, iy, iz, gridSideCount);

        // fill dev_particleGridIndices at boidIdx
        gridIndices[boidIdx] = cellIdx;

        // fill dev_particleArrayIndices, starts exactly as boidIdx initially.
        indices[boidIdx] = boidIdx;
    }

}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
// pass in an invalid array value such as -1 for cells that don't contain boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"

    // post sorting
    // also looks like it's called in parallel by boids.
    // note that idx is NOT boidIdx
    // for current idx, check its own cell in GridIndices and one before it.
    // if idx 0, must be start cell, and set its start location to itself. then check for end cell status
    // if idx N - 1, must be end cell, and set its end location to itself. then check for start cell status
    // otherwise, if different from idx - 1, set as start cell.
    // if different from idx + 1, set as end cell.
    // not sure if most efficient way to do it honestly.

    int accessIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (accessIdx < N) {
        int gridCell = particleGridIndices[accessIdx];

        if (accessIdx == 0) {
            gridCellStartIndices[gridCell] = accessIdx;

            // check for next grid cell as well
            int nextGridCell = particleGridIndices[accessIdx + 1];
            if (gridCell != nextGridCell) {
                gridCellEndIndices[gridCell] = accessIdx;
            }
        }
        else if (accessIdx == N - 1) {
            gridCellEndIndices[gridCell] = accessIdx;

            // check if gridCell == prevGridCell. 
            int prevGridCell = particleGridIndices[accessIdx - 1];
            if (gridCell != prevGridCell) {
                gridCellStartIndices[gridCell] = accessIdx;
            }
        }
        else {
            int prevGridCell = particleGridIndices[accessIdx - 1];
            int nextGridCell = particleGridIndices[accessIdx + 1];
            if (gridCell != prevGridCell) {
                gridCellStartIndices[gridCell] = accessIdx;
            }
            if (gridCell != nextGridCell) {
                gridCellEndIndices[gridCell] = accessIdx;
            }
        }
    }
}

// device helper function
__device__ glm::vec3 kernComputeVelocityChangeScattered(int N, glm::vec3 gridMin, 
    float cellWidth, int iSelf, glm::vec3* pos, 
    glm::vec3* vel1, int* gridCellStartIndices, 
    int* gridCellEndIndices, int* particleArrayIndices, int sideCount) {

    int selfIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (selfIdx < N) {
        glm::vec3 selfPos = pos[selfIdx];
        glm::vec3 velocityChange = glm::vec3(0.f);

        float radius = imax(imax(rule1Distance, rule2Distance), rule3Distance);

        int numValid1 = 0;
        int numValid3 = 0;

        glm::vec3 center = glm::vec3(0.f);
        glm::vec3 c = glm::vec3(0.f);
        glm::vec3 perceivedVelocity = glm::vec3(0.f);

        // find maximum of entire grid in cell space coordinates.
        glm::vec3 cellSpaceMax = glm::vec3(sideCount - 1, sideCount - 1, sideCount - 1);

        // create a bounding box based on the location of boid within the grid in cell space coordinates.
        float xMax, xMin, yMax, yMin, zMax, zMin = 0;

        // xMax and co. are all cell space coordinates and not world space coordaintes.
        // find the grid cell associated with radMaxx, radMaxy, radMaxz, radMinx, radMiny, radMinz
        // set xMax, xMin, yMax, yMin, zMax, zMin etc by clamping between that and 
        int ixRadMax = std::floor((selfPos.x + radius - gridMin.x) / cellWidth);
        int iyRadMax = std::floor((selfPos.y + radius - gridMin.y) / cellWidth);
        int izRadMax = std::floor((selfPos.z + radius - gridMin.z) / cellWidth);
        int ixRadMin = std::floor((selfPos.x - radius - gridMin.x) / cellWidth);
        int iyRadMin = std::floor((selfPos.y - radius - gridMin.y) / cellWidth);
        int izRadMin = std::floor((selfPos.z - radius - gridMin.z) / cellWidth);

        xMax = imin(ixRadMax, cellSpaceMax.x);
        yMax = imin(iyRadMax, cellSpaceMax.y);
        zMax = imin(izRadMax, cellSpaceMax.z);
        xMin = imax(ixRadMin, gridMin.x);
        yMin = imax(iyRadMin, gridMin.y);
        zMin = imax(izRadMin, gridMin.z);

        // loop within the bounding box.
        // these coordinates have already been clamped
        // without consideration of which axis is best to check first
        // started with x,y,z

        for (float z = zMin; z <= zMax; z += 1) {
            for (float y = yMin; y <= yMax; y += 1) {
                for (float x = xMin; x <= xMax; x += 1) {
                    int gridIdx = gridIndex3Dto1D(x, y, z, sideCount);
                    int startIdx = gridCellStartIndices[gridIdx];
                    int endIdx = gridCellEndIndices[gridIdx];

                    if (startIdx > -1) {
                        // if start idx is > -1, then boids exist
                        for (int curIdx = startIdx; curIdx <= endIdx; curIdx++) {
                            int nBoidIdx = particleArrayIndices[curIdx];
                            glm::vec3 nBoidPos = pos[nBoidIdx];

                            float dist = glm::distance(selfPos, nBoidPos);

                            // Rule 1
                            if (nBoidIdx != selfIdx && dist < rule1Distance) {
                                center += pos[nBoidIdx];
                                numValid1 += 1;
                            }

                            // Rule 2
                            if (nBoidIdx != selfIdx && dist < rule2Distance) {
                                c -= (nBoidPos - selfPos);
                            }

                            // Rule 3
                            if (nBoidIdx != selfIdx && dist < rule3Distance) {
                                perceivedVelocity += vel1[nBoidIdx];
                                numValid3 += 1;
                            }
                        }
                    }
                }
            }
        }

        if (numValid1 != 0) {
            center /= numValid1;
            velocityChange += (center - selfPos) * rule1Scale;
        }

        velocityChange += c * rule2Scale;

        if (numValid3 != 0) {
            perceivedVelocity /= numValid3;
            velocityChange += perceivedVelocity * rule3Scale;
        }

        return velocityChange;
    }
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridSideCount, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2

    // for a given boid, try all cells first, then optimize by doing quadrant search later.
    // 1. split cell into octants, identify the octant it belongs in
    // 2. get 8 neighboring octants. figure out which octants contain cells. --> get all filled cells regardless of distance
    // 2. Scan the appropriate 8 or so octants based on the result
    // 3. for each cell, read the start/end endices and read boid in each cell. apply velocity changes.
    // 4. clamp speed as usual

    int selfIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (selfIdx < N) {
        glm::vec3 velocityChange = kernComputeVelocityChangeScattered(N, gridMin, cellWidth, selfIdx, pos, vel1, gridCellStartIndices, 
            gridCellEndIndices, particleArrayIndices, gridSideCount);

        // clamp speed and set new velocity
        glm::vec3 finalVel = vel1[selfIdx] + velocityChange;
        float speed = glm::length(finalVel);
        if (speed > maxSpeed) {
            finalVel = finalVel * maxSpeed / speed;
        }
        // printf("finalVel: x: %f, y: %f, z: %f \n", finalVel.x, finalVel.y, finalVel.z);

        vel2[selfIdx] = finalVel;
    }
}

// helper device
__device__ glm::vec3 kernComputeVelocityChangeCoherent(int N, glm::vec3 gridMin,
    float cellWidth, int iSelf, glm::vec3* posSorted,
    glm::vec3* vel1Sorted, int* gridCellStartIndices,
    int* gridCellEndIndices, int sideCount) {
    // int selfIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (iSelf < N) {
        glm::vec3 selfPos = posSorted[iSelf];
        glm::vec3 velocityChange = glm::vec3(0.f);

        float radius = imax(imax(rule1Distance, rule2Distance), rule3Distance);

        int numValid1 = 0;
        int numValid3 = 0;

        glm::vec3 center = glm::vec3(0.f);
        glm::vec3 c = glm::vec3(0.f);
        glm::vec3 perceivedVelocity = glm::vec3(0.f);

        // find maximum of entire grid in cell space coordinates.
        glm::vec3 cellSpaceMax = glm::vec3(sideCount - 1, sideCount - 1, sideCount - 1);

        // create a bounding box based on the location of boid within the grid in cell space coordinates.
        float xMax, xMin, yMax, yMin, zMax, zMin = 0;

        // xMax and co. are all cell space coordinates and not world space coordaintes.
        // find the grid cell associated with radMaxx, radMaxy, radMaxz, radMinx, radMiny, radMinz
        // set xMax, xMin, yMax, yMin, zMax, zMin etc by clamping between that and 
        int ixRadMax = std::floor((selfPos.x + radius - gridMin.x) / cellWidth);
        int iyRadMax = std::floor((selfPos.y + radius - gridMin.y) / cellWidth);
        int izRadMax = std::floor((selfPos.z + radius - gridMin.z) / cellWidth);
        int ixRadMin = std::floor((selfPos.x - radius - gridMin.x) / cellWidth);
        int iyRadMin = std::floor((selfPos.y - radius - gridMin.y) / cellWidth);
        int izRadMin = std::floor((selfPos.z - radius - gridMin.z) / cellWidth);

        xMax = imin(ixRadMax, cellSpaceMax.x);
        yMax = imin(iyRadMax, cellSpaceMax.y);
        zMax = imin(izRadMax, cellSpaceMax.z);
        xMin = imax(ixRadMin, gridMin.x);
        yMin = imax(iyRadMin, gridMin.y);
        zMin = imax(izRadMin, gridMin.z);

        // loop within the bounding box.
        // these coordinates have already been clamped
        // todo consider which axis is the best for checking
        for (float x = xMin; x <= xMax; x += 1) {
            for (float y = yMin; y <= yMax; y += 1) {
                for (float z = zMin; z <= zMax; z += 1) {
                    int gridIdx = gridIndex3Dto1D(x, y, z, sideCount);
                    int startIdx = gridCellStartIndices[gridIdx];
                    int endIdx = gridCellEndIndices[gridIdx];

                    if (startIdx > -1) {
                        // if start idx is > -1, then boids exist
                        for (int curIdx = startIdx; curIdx <= endIdx; curIdx++) {
                            // now curIdx is the same as nBoidIdx
                            glm::vec3 nBoidPos = posSorted[curIdx];
                            glm::vec3 vel = vel1Sorted[curIdx];

                            float dist = glm::distance(selfPos, nBoidPos);
                            //printf("dist: %f \n", dist);

                            // Rule 1
                            if (curIdx != iSelf && dist < rule1Distance) {
                                center += posSorted[curIdx];
                                numValid1 += 1;
                            }

                            // Rule 2
                            if (curIdx != iSelf && dist < rule2Distance) {
                                c -= (nBoidPos - selfPos);
                            }

                            // Rule 3
                            if (curIdx != iSelf && dist < rule3Distance) {
                                perceivedVelocity += vel1Sorted[curIdx];
                                numValid3 += 1;
                            }
                        }
                    }
                }
            }
        }

        if (numValid1 != 0) {
            center /= numValid1;
            velocityChange += (center - selfPos) * rule1Scale;
        }

        velocityChange += c * rule2Scale;

        if (numValid3 != 0) {
            perceivedVelocity /= numValid3;
            velocityChange += perceivedVelocity * rule3Scale;
        }

        return velocityChange;
    }
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridSideCount, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *posSorted, glm::vec3 *vel1Sorted, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
    int selfIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (selfIdx < N) {
        glm::vec3 velocityChange = kernComputeVelocityChangeCoherent(N, gridMin, 
            cellWidth, selfIdx, posSorted, 
            vel1Sorted, gridCellStartIndices,
            gridCellEndIndices, gridSideCount);

        // clamp speed and set new velocity
        glm::vec3 finalVel = vel1Sorted[selfIdx] + velocityChange;
        float speed = glm::length(finalVel);
        if (speed > maxSpeed) {
            finalVel = finalVel * maxSpeed / speed;
        }
        // printf("finalVel: x: %f, y: %f, z: %f \n", finalVel.x, finalVel.y, finalVel.z);

        // vel2 is still the write buffer
        vel2[selfIdx] = finalVel;
    }
}

// helper global which rearranges information in pos and vel based on particleArrayIndices O(N) time

__global__ void kernRearrangePosVel(int N, int* particleArrayIndices, glm::vec3* posSorted, glm::vec3* vel1Sorted, glm::vec3* pos, glm::vec3* vel1) {
    int selfIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (selfIdx < N) {
        int requiredIdx = particleArrayIndices[selfIdx];

        //printf("vel x: %f, y: %f, z: %f \n ", vel1Prev[requiredIdx].x, vel1Prev[requiredIdx].y, vel1Prev[requiredIdx].z);

        // essentially swap the contents
        glm::vec3 requiredPos = pos[requiredIdx];
        glm::vec3 requiredVel = vel1[requiredIdx];

        posSorted[selfIdx] = requiredPos;
        vel1Sorted[selfIdx] = requiredVel;
    }
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
  // TODO-1.2 ping-pong the velocity buffers
    int fullBlocksPerGrid = (numObjects + blockSize - 1) / blockSize;

    kernUpdateVelocityBruteForce<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, dev_vel1, dev_vel2);
    kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel1);

    std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed
    int numCells = gridSideCount * gridSideCount * gridSideCount;

    // For each boid, label its array index and grid index
    int fullBlocksPerGrid = (numObjects + blockSize - 1) / blockSize;
    int fullCellsPerGrid = (numCells + blockSize - 1) / blockSize;

    // fill with -1s as default values
    // Per cell basis
    kernResetIntBuffer<<<fullCellsPerGrid, blockSize>>>(numCells, dev_gridCellStartIndices, -1);
    kernResetIntBuffer<<<fullCellsPerGrid, blockSize>>>(numCells, dev_gridCellEndIndices, -1);
    
    // per boid basis
    kernComputeIndices<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth,
        dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

    // pointer to first key, pointer to last key, pointer to first value
    thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

    kernIdentifyCellStartEnd<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

    // per boid basis
    kernUpdateVelNeighborSearchScattered << <fullBlocksPerGrid, blockSize >> >(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth,
        dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);

    // update position per boid
    kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel1);

    // ping pong
    std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
    int numCells = gridSideCount * gridSideCount * gridSideCount;

    // For each boid, label its array index and grid index
    int fullBlocksPerGrid = (numObjects + blockSize - 1) / blockSize;
    int fullCellsPerGrid = (numCells + blockSize - 1) / blockSize;

    // fill with -1s as default values
    // Per cell basis
    kernResetIntBuffer << <fullCellsPerGrid, blockSize >> > (numCells, dev_gridCellStartIndices, -1);
    kernResetIntBuffer << <fullCellsPerGrid, blockSize >> > (numCells, dev_gridCellEndIndices, -1);

    // per boid basis
    kernComputeIndices << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth,
        dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

    // sort dev_array_indices values
    thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

    // rearrange pos and vel
    // we never change vel1 and pos. only use them as reads.
    kernRearrangePosVel << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleArrayIndices, dev_posSorted, dev_vel1Sorted, dev_pos, dev_vel1);

    // find start and end indices
// per boid basis because we read from dev_particleGridIndices
    kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

    // per boid basis  
    kernUpdateVelNeighborSearchCoherent << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum, 
        gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, 
        dev_gridCellEndIndices, dev_posSorted, dev_vel1Sorted, dev_vel2);

    // update position per boid
    kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt, dev_posSorted, dev_vel2);

    // ping pong
    std::swap(dev_vel1, dev_vel2);
    std::swap(dev_posSorted, dev_pos);
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);

  // cannot free the thrust pointers
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU

  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
