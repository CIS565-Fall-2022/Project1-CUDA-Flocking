#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char* msg, int line = -1) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        if (line >= 0) {
            fprintf(stderr, "Line %d: ", line);
        }
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3* dev_pos;
glm::vec3* dev_vel1;
glm::vec3* dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int* dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int* dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int* dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int* dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
thrust::device_ptr<glm::vec3> dev_thrust_particlePos;
thrust::device_ptr<glm::vec3> dev_thrust_particleVel;

//array index copy
thrust::device_ptr<int> dev_thrust_particleArrayIndices2;
int* dev_particleArrayIndices2;
thrust::device_ptr<int> dev_thrust_particleGridIndices2;
int* dev_particleGridIndices2;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
    a = (a + 0x7ed55d16) + (a << 12);
    a = (a ^ 0xc761c23c) ^ (a >> 19);
    a = (a + 0x165667b1) + (a << 5);
    a = (a + 0xd3a2646c) ^ (a << 9);
    a = (a + 0xfd7046c5) + (a << 3);
    a = (a ^ 0xb55a4f09) ^ (a >> 16);
    return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
    thrust::default_random_engine rng(hash((int)(index * time)));
    thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

    return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3* arr, float scale) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < N) {
        glm::vec3 rand = generateRandomVec3(time, index);
        arr[index].x = scale * rand.x;
        arr[index].y = scale * rand.y;
        arr[index].z = scale * rand.z;
    }
}

__global__ void kernGenerateZeroedVelArray(int time, int N, glm::vec3* arr) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < N) {
        arr[index].x = 0;
        arr[index].y = 0;
        arr[index].z = 0;
    }
}

__global__ void kernGenerateRandomVelArray(int time, int N, glm::vec3* arr) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < N) {
        glm::vec3 rand = generateRandomVec3(time, index);
        arr[index].x =  rand.x;
        arr[index].y =  rand.y;
        arr[index].z =  rand.z;
    }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
    numObjects = N;
    dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

    // LOOK-1.2 - This is basic CUDA memory management and error checking.
    // Don't forget to hipFree in  Boids::endSimulation.
    hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
    checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

    hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
    checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

    hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
    checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

    // LOOK-1.2 - This is a typical CUDA kernel invocation.
    kernGenerateRandomPosArray << <fullBlocksPerGrid, blockSize >> > (1, numObjects,
        dev_pos, scene_scale);
    checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

    /*kernGenerateRandomVelArray << <fullBlocksPerGrid, blockSize >> > (2, numObjects,
        dev_vel1);
    checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");*/

    // LOOK-2.1 computing grid params
    gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
    int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
    gridSideCount = 2 * halfSideCount;

    gridCellCount = gridSideCount * gridSideCount * gridSideCount;
    gridInverseCellWidth = 1.0f / gridCellWidth;
    float halfGridWidth = gridCellWidth * halfSideCount;
    gridMinimum.x -= halfGridWidth;
    gridMinimum.y -= halfGridWidth;
    gridMinimum.z -= halfGridWidth;

    // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
    hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
    checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

    hipMalloc((void**)&dev_particleArrayIndices2, N * sizeof(int));
    checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

    hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
    checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

    hipMalloc((void**)&dev_particleGridIndices2, N * sizeof(int));
    checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

    hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
    checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

    hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
    checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

    hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3* pos, float* vbo, float s_scale) {
    int index = threadIdx.x + (blockIdx.x * blockDim.x);

    float c_scale = -1.0f / s_scale;

    if (index < N) {
        vbo[4 * index + 0] = pos[index].x * c_scale;
        vbo[4 * index + 1] = pos[index].y * c_scale;
        vbo[4 * index + 2] = pos[index].z * c_scale;
        vbo[4 * index + 3] = 1.0f;
    }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3* vel, float* vbo, float s_scale) {
    int index = threadIdx.x + (blockIdx.x * blockDim.x);

    if (index < N) {
        vbo[4 * index + 0] = vel[index].x + 0.3f;
        vbo[4 * index + 1] = vel[index].y + 0.3f;
        vbo[4 * index + 2] = vel[index].z + 0.3f;
        vbo[4 * index + 3] = 1.0f;
    }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float* vbodptr_positions, float* vbodptr_velocities) {
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

    kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_pos, vbodptr_positions, scene_scale);
    kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_vel1, vbodptr_velocities, scene_scale);

    checkCUDAErrorWithLine("copyBoidsToVBO failed!");

    hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3* pos, const glm::vec3* vel) {
    // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
    // Rule 2: boids try to stay a distance d away from each other
    // Rule 3: boids try to match the speed of surrounding boids
    int neighboors = 0;
    float center_x = 0.0;
    float center_y = 0.0;
    float center_z = 0.0;

    float separate_x = 0.0;
    float separate_y = 0.0;
    float separate_z = 0.0;

    float cohesion_x = 0.0;
    float cohesion_y = 0.0;
    float cohesion_z = 0.0;


    for (int boid = 0; boid < N; boid++) {
        if (boid == iSelf) continue;
        float distance = glm::distance(pos[iSelf], pos[boid]);
        if (distance < rule1Distance) {
            center_x += pos[boid].x;
            center_y += pos[boid].y;
            center_z += pos[boid].z;
            neighboors++;

            if (distance < rule2Distance) {
                separate_x -= pos[boid].x - pos[iSelf].x;
                separate_y -= pos[boid].y - pos[iSelf].y;
                separate_z -= pos[boid].z - pos[iSelf].z;
            }

            cohesion_x += vel[boid].x;
            cohesion_y += vel[boid].y;
            cohesion_z += vel[boid].z;
        }
    }

    glm::vec3 retVec = vel[iSelf];
    glm::vec3 centerVec = glm::vec3(center_x, center_y, center_z);
    glm::vec3 separateVec = glm::vec3(separate_x, separate_y, separate_z);
    glm::vec3 cohesionVec = glm::vec3(cohesion_x, cohesion_y, cohesion_z);

    if (neighboors > 0) {
        centerVec /= neighboors;
        retVec += (centerVec - pos[iSelf]) * rule1Scale;
        retVec += cohesionVec * rule3Scale;
    }
    retVec += separateVec * rule2Scale;
    return retVec;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3* pos,
    glm::vec3* vel1, glm::vec3* vel2) {
    // Compute a new velocity based on pos and vel1
    // Clamp the speed
    // Record the new velocity into vel2. Question: why NOT vel1?
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) {
        return;
    }
    glm::vec3 v = computeVelocityChange(N, index, pos, vel1);
    float vSize = glm::length(v);
    if (vSize > maxSpeed) {
        v = v / vSize * maxSpeed;
    }
    vel2[index] = v;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3* pos, glm::vec3* vel) {
    // Update position by velocity
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) {
        return;
    }
    glm::vec3 thisPos = pos[index];
    thisPos += vel[index] * dt;

    // Wrap the boids around so we don't lose them
    thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
    thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
    thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

    thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
    thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
    thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

    pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
    return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
    glm::vec3 gridMin, float inverseCellWidth,
    glm::vec3* pos, int* indices, int* gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < N) {
        glm::vec3 relativePos = pos[index] - gridMin;
        relativePos *= inverseCellWidth;
        gridIndices[index] = gridIndex3Dto1D(relativePos.x, relativePos.y, relativePos.z, gridResolution);
        indices[index] = index;
    }
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int* intBuffer, int value) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < N) {
        intBuffer[index] = value;
    }
}

__global__ void kernIdentifyCellStartEnd(int N, int* particleGridIndices,
    int* gridCellStartIndices, int* gridCellEndIndices) {
    // TODO-2.1
    // Identify the start point of each cell in the gridIndices array.
    // This is basically a parallel unrolling of a loop that goes
    // "this index doesn't match the one before it, must be a new cell!"
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < N) {
        int gridThis = particleGridIndices[index];
        if (index == 0) {
            gridCellStartIndices[gridThis] = index;
        }
        else {
            if (index == N - 1) {
                gridCellEndIndices[gridThis] = index;
            }
            int gridLast = particleGridIndices[index - 1];
            if (gridLast != gridThis) {
                gridCellStartIndices[gridThis] = index;
                gridCellEndIndices[gridLast] = index - 1;
            }
        }
    }
}

__global__ void kernUpdateVelNeighborSearchScattered(
    int N, int gridResolution, glm::vec3 gridMin,
    float inverseCellWidth, float cellWidth,
    int* gridCellStartIndices, int* gridCellEndIndices,
    int* particleArrayIndices,
    glm::vec3* pos, glm::vec3* vel1, glm::vec3* vel2) {
    // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
    // the number of boids that need to be checked.
    // - Identify the grid cell that this particle is in
    // - Identify which cells may contain neighbors. This isn't always 8.
    // - For each cell, read the start/end indices in the boid pointer array.
    // - Access each boid in the cell and compute velocity change from
    //   the boids rules, if this boid is within the neighborhood distance.
    // - Clamp the speed change before putting the new speed in vel2
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) {
        return;
    }

    //The max check distancce is equal to half of cell width, as defined by init simulation
    glm::vec3 minPoint = pos[index] - cellWidth / 2;
    glm::vec3 retVec = vel1[index];
  
    int neighboors = 0;
    float center_x = 0.0;
    float center_y = 0.0;
    float center_z = 0.0;

    float separate_x = 0.0;
    float separate_y = 0.0;
    float separate_z = 0.0;

    float cohesion_x = 0.0;
    float cohesion_y = 0.0;
    float cohesion_z = 0.0;

    //The 3/2 is just a jank way of getting rid of floating point errors
    for (int z = minPoint.z; z <= minPoint.z + cellWidth * 3 / 2; z += cellWidth) {
        for (int y = minPoint.y; y <= minPoint.y + cellWidth * 3 / 2; y += cellWidth) {
            for (int x = minPoint.x; x <= minPoint.x + cellWidth * 3 / 2; x += cellWidth) {
                glm::vec3 relativePos = glm::vec3(x, y, z) - gridMin;
                relativePos *= inverseCellWidth;
                int cell = gridIndex3Dto1D(relativePos.x, relativePos.y, relativePos.z, gridResolution);
                if (relativePos.x < 0 || relativePos.y < 0 || relativePos.z < 0 
                    || relativePos.x >= gridResolution || relativePos.y >= gridResolution || relativePos.z >= gridResolution
                    || gridCellStartIndices[cell] < 0) {
                    continue;
                }               
                for (int idx = gridCellStartIndices[cell]; idx <= gridCellEndIndices[cell]; idx++) {              
                    int boid = particleArrayIndices[idx];
                    
                    if (boid == index) continue;
                    float distance = glm::distance(pos[index], pos[boid]);
                    if (distance < rule1Distance) {
                        center_x += pos[boid].x;
                        center_y += pos[boid].y;
                        center_z += pos[boid].z;
                        neighboors++;

                        if (distance < rule2Distance) {
                            separate_x -= pos[boid].x - pos[index].x;
                            separate_y -= pos[boid].y - pos[index].y;
                            separate_z -= pos[boid].z - pos[index].z;
                        }

                        cohesion_x += vel1[boid].x;
                        cohesion_y += vel1[boid].y;
                        cohesion_z += vel1[boid].z;
                    }
                }
            }
        }
    }
    
    glm::vec3 centerVec = glm::vec3(center_x, center_y, center_z);
    glm::vec3 separateVec = glm::vec3(separate_x, separate_y, separate_z);
    glm::vec3 cohesionVec = glm::vec3(cohesion_x, cohesion_y, cohesion_z);

    if (neighboors > 0) {
        centerVec /= neighboors;
        retVec += (centerVec - pos[index]) * rule1Scale;
        retVec += cohesionVec * rule3Scale;
    }
    retVec += separateVec * rule2Scale;
    float vSize = glm::length(retVec);
    if (vSize > maxSpeed) {
        retVec = retVec / vSize * maxSpeed;
    }
    vel2[index] = retVec;
    
}

__global__ void kernUpdateVelNeighborSearchCoherent(
    int N, int gridResolution, glm::vec3 gridMin,
    float inverseCellWidth, float cellWidth,
    int* gridCellStartIndices, int* gridCellEndIndices,
    glm::vec3* pos, glm::vec3* vel1, glm::vec3* vel2) {
    // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
    // except with one less level of indirection.
    // This should expect gridCellStartIndices and gridCellEndIndices to refer
    // directly to pos and vel1.
    // - Identify the grid cell that this particle is in
    // - Identify which cells may contain neighbors. This isn't always 8.
    // - For each cell, read the start/end indices in the boid pointer array.
    //   DIFFERENCE: For best results, consider what order the cells should be
    //   checked in to maximize the memory benefits of reordering the boids data.
    // - Access each boid in the cell and compute velocity change from
    //   the boids rules, if this boid is within the neighborhood distance.
    // - Clamp the speed change before putting the new speed in vel2
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) {
        return;
    }
    //The max check distancce is equal to half of cell width, as defined by init simulation
    glm::vec3 minPoint = pos[index] - cellWidth / 2;
    glm::vec3 retVec = vel1[index];

    int neighboors = 0;
    float center_x = 0.0;
    float center_y = 0.0;
    float center_z = 0.0;

    float separate_x = 0.0;
    float separate_y = 0.0;
    float separate_z = 0.0;

    float cohesion_x = 0.0;
    float cohesion_y = 0.0;
    float cohesion_z = 0.0;

    //The 3/2 is just a jank way of getting rid of floating point errors
    for (int z = minPoint.z; z <= minPoint.z + cellWidth * 3 / 2; z += cellWidth) {
        for (int y = minPoint.y; y <= minPoint.y + cellWidth * 3 / 2; y += cellWidth) {
            for (int x = minPoint.x; x <= minPoint.x + cellWidth * 3 / 2; x += cellWidth) {
                glm::vec3 relativePos = glm::vec3(x, y, z) - gridMin;
                relativePos *= inverseCellWidth;
                int cell = gridIndex3Dto1D(relativePos.x, relativePos.y, relativePos.z, gridResolution);
                if (relativePos.x < 0 || relativePos.y < 0 || relativePos.z < 0
                    || relativePos.x >= gridResolution || relativePos.y >= gridResolution || relativePos.z >= gridResolution
                    || gridCellStartIndices[cell] < 0) {
                    continue;
                }
                for (int idx = gridCellStartIndices[cell]; idx <= gridCellEndIndices[cell]; idx++) {
                    int boid = idx;

                    if (boid == index) continue;
                    float distance = glm::distance(pos[index], pos[boid]);
                    if (distance < rule1Distance) {
                        center_x += pos[boid].x;
                        center_y += pos[boid].y;
                        center_z += pos[boid].z;
                        neighboors++;

                        if (distance < rule2Distance) {
                            separate_x -= pos[boid].x - pos[index].x;
                            separate_y -= pos[boid].y - pos[index].y;
                            separate_z -= pos[boid].z - pos[index].z;
                        }

                        cohesion_x += vel1[boid].x;
                        cohesion_y += vel1[boid].y;
                        cohesion_z += vel1[boid].z;
                    }
                }
            }
        }
    }

    glm::vec3 centerVec = glm::vec3(center_x, center_y, center_z);
    glm::vec3 separateVec = glm::vec3(separate_x, separate_y, separate_z);
    glm::vec3 cohesionVec = glm::vec3(cohesion_x, cohesion_y, cohesion_z);

    if (neighboors > 0) {
        centerVec /= neighboors;
        retVec += (centerVec - pos[index]) * rule1Scale;
        retVec += cohesionVec * rule3Scale;
    }
    retVec += separateVec * rule2Scale;
    float vSize = glm::length(retVec);
    if (vSize > maxSpeed) {
        retVec = retVec / vSize * maxSpeed;
    }
    vel2[index] = retVec;
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
    // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
    // TODO-1.2 ping-pong the velocity buffers
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
    kernUpdateVelocityBruteForce << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_pos, dev_vel1, dev_vel2);
    checkCUDAErrorWithLine("kernUpdateVelocityBruteForce failed!");
    kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt, dev_pos, dev_vel2);
    checkCUDAErrorWithLine("kernUpdatePos failed!");
    glm::vec3* temp = dev_vel1;
    dev_vel1 = dev_vel2;
    dev_vel2 = temp;
}

void Boids::stepSimulationScatteredGrid(float dt) {
    // TODO-2.1
    // Uniform Grid Neighbor search using Thrust sort.
    // In Parallel:
    // - label each particle with its array index as well as its grid index.
    //   Use 2x width grids.
    // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
    //   are welcome to do a performance comparison.
    // - Naively unroll the loop for finding the start and end indices of each
    //   cell's data pointers in the array of boid indices
    // - Perform velocity updates using neighbor search
    // - Update positions
    // - Ping-pong buffers as needed
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
    kernComputeIndices << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
    checkCUDAErrorWithLine("kernComputeIndices failed!");

    thrust::device_ptr<int> dev_thrust_particleArrayIndices(dev_particleArrayIndices);
    thrust::device_ptr<int> dev_thrust_particleGridIndices(dev_particleGridIndices);
    // MIGHT HAVE BUGS LOOK HERE DUMBASS
    thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

    kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_gridCellStartIndices, -1);
    checkCUDAErrorWithLine("kernResetIntBuffer for start failed!");

    kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_gridCellEndIndices, -1);
    checkCUDAErrorWithLine("kernResetIntBuffer for end failed!");

    kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
    checkCUDAErrorWithLine("kernIdentifyCellStartEnd for end failed!");

    kernUpdateVelNeighborSearchScattered << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum,
        gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices,
        dev_pos, dev_vel1, dev_vel2);
    checkCUDAErrorWithLine("kernUpdateVelNeighborSearchScattered for end failed!");

    kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt, dev_pos, dev_vel2);
    checkCUDAErrorWithLine("kernUpdatePos failed!");

    glm::vec3* temp = dev_vel1;
    dev_vel1 = dev_vel2;
    dev_vel2 = temp;
}

void Boids::stepSimulationCoherentGrid(float dt) {
    // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
    // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
    // In Parallel:
    // - Label each particle with its array index as well as its grid index.
    //   Use 2x width grids
    // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
    //   are welcome to do a performance comparison.
    // - Naively unroll the loop for finding the start and end indices of each
    //   cell's data pointers in the array of boid indices
    // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
    //   the particle data in the simulation array.
    //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
    // - Perform velocity updates using neighbor search
    // - Update positions
    // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
    kernComputeIndices << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
    checkCUDAErrorWithLine("kernComputeIndices failed!");

    thrust::device_ptr<int> dev_thrust_particleArrayIndices(dev_particleArrayIndices);
    thrust::device_ptr<int> dev_thrust_particleGridIndices(dev_particleGridIndices);
    thrust::device_ptr<int> dev_thrust_particleGridIndices2(dev_particleGridIndices2);
    //thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);
    thrust::device_ptr<glm::vec3> dev_thrust_particlePos(dev_pos);
    thrust::device_ptr<glm::vec3> dev_thrust_particleVel(dev_vel1);
    hipMemcpy(dev_particleGridIndices2, dev_particleGridIndices, sizeof(int) * numObjects, hipMemcpyDeviceToDevice);
    thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particlePos);
    thrust::sort_by_key(dev_thrust_particleGridIndices2, dev_thrust_particleGridIndices2 + numObjects, dev_thrust_particleVel);
   

    //hipMemcpy(dev_particleArrayIndices2, dev_particleArrayIndices, sizeof(int) * numObjects, hipMemcpyDeviceToDevice);
    //thrust::device_ptr<int>dev_thrust_particleArrayIndices2(dev_particleArrayIndices2);

    kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_gridCellStartIndices, -1);
    checkCUDAErrorWithLine("kernResetIntBuffer for start failed!");

    kernResetIntBuffer << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_gridCellEndIndices, -1);
    checkCUDAErrorWithLine("kernResetIntBuffer for end failed!");

    kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
    checkCUDAErrorWithLine("kernIdentifyCellStartEnd for end failed!");


    kernUpdateVelNeighborSearchCoherent << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum,
        gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, 
        dev_pos, dev_vel1, dev_vel2);
    checkCUDAErrorWithLine("kernUpdateVelNeighborSearchScattered for end failed!");

    kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt, dev_pos, dev_vel2);
    checkCUDAErrorWithLine("kernUpdatePos failed!");

    glm::vec3* temp = dev_vel1;
    dev_vel1 = dev_vel2;
    dev_vel2 = temp;
}

void Boids::endSimulation() {
    hipFree(dev_vel1);
    hipFree(dev_vel2);
    hipFree(dev_pos);
    hipFree(dev_particleArrayIndices);
    hipFree(dev_particleArrayIndices2);
    hipFree(dev_particleGridIndices);
    hipFree(dev_particleGridIndices2);
    hipFree(dev_gridCellStartIndices);
    hipFree(dev_gridCellEndIndices);

    // TODO-2.1 TODO-2.3 - Free any additional buffers here.
}

void Boids::unitTest() {
    // LOOK-1.2 Feel free to write additional tests here.

    // test unstable sort
    int* dev_intKeys;
    int* dev_intValues;
    int N = 10;

    std::unique_ptr<int[]>intKeys{ new int[N] };
    std::unique_ptr<int[]>intValues{ new int[N] };

    std::unique_ptr<int[]>intGrid{ new int[numObjects] };
    std::unique_ptr<int[]>intArr{ new int[numObjects] };
    std::unique_ptr<int[]>intStart{ new int[gridCellCount] };
    std::unique_ptr<int[]>intEnd{ new int[gridCellCount] };
    std::unique_ptr<glm::vec3[]>posV{ new glm::vec3[gridCellCount] };
    std::unique_ptr<glm::vec3[]>vel1{ new glm::vec3[gridCellCount] };
    std::unique_ptr<glm::vec3[]>vel2{ new glm::vec3[gridCellCount] };
    std::unique_ptr<glm::vec3[]>posVNaive{ new glm::vec3[gridCellCount] };

    intKeys[0] = 0; intValues[0] = 0;
    intKeys[1] = 1; intValues[1] = 1;
    intKeys[2] = 0; intValues[2] = 2;
    intKeys[3] = 3; intValues[3] = 3;
    intKeys[4] = 0; intValues[4] = 4;
    intKeys[5] = 2; intValues[5] = 5;
    intKeys[6] = 2; intValues[6] = 6;
    intKeys[7] = 0; intValues[7] = 7;
    intKeys[8] = 5; intValues[8] = 8;
    intKeys[9] = 6; intValues[9] = 9;

    hipMalloc((void**)&dev_intKeys, N * sizeof(int));
    checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

    hipMalloc((void**)&dev_intValues, N * sizeof(int));
    checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

    dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

    /*std::cout << "before unstable sort: " << std::endl;
    for (int i = 0; i < N; i++) {
        std::cout << "  key: " << intKeys[i];
        std::cout << " value: " << intValues[i] << std::endl;
    }*/
    /*stepSimulationNaive(.2f);
    hipMemcpy(posVNaive.get(), dev_pos, sizeof(glm::vec3) * numObjects, hipMemcpyDeviceToHost);
    for (int i = 0; i < 5000; i++) {
        std::cout << " x: " << posVNaive[i].x << " y: " << posVNaive[i].y << " z: " << posVNaive[i].z << std::endl;
    }*/
    //stepSimulationCoherentGrid(.2f);

    hipMemcpy(intGrid.get(), dev_particleGridIndices, sizeof(int) * numObjects, hipMemcpyDeviceToHost);
    hipMemcpy(intArr.get(), dev_particleArrayIndices, sizeof(int) * numObjects, hipMemcpyDeviceToHost);
    hipMemcpy(intStart.get(), dev_gridCellStartIndices, sizeof(int) * gridCellCount, hipMemcpyDeviceToHost);
    hipMemcpy(intEnd.get(), dev_gridCellEndIndices, sizeof(int) * gridCellCount, hipMemcpyDeviceToHost);
    hipMemcpy(posV.get(), dev_pos, sizeof(glm::vec3) * numObjects, hipMemcpyDeviceToHost);
    hipMemcpy(vel1.get(), dev_vel1, sizeof(glm::vec3) * numObjects, hipMemcpyDeviceToHost);
    hipMemcpy(vel2.get(), dev_vel2, sizeof(glm::vec3) * numObjects, hipMemcpyDeviceToHost);
    
    //std::cout << "cell width: " << gridCellWidth << " cell resolution: " << gridSideCount << " ------------------------" << std::endl;
    //for (int i = 0; i < 5000; i++) {
    //    
    //    std::cout << "  key: " << intGrid[i];
    //    //std::cout << " value: " << intArr[i];
    //    std::cout << " start: " << intStart[intGrid[i]] << " end: " << intEnd[intGrid[i]] << std::endl;
    //    std::cout << " x: " << posV[i].x << " y: " << posV[i].y << " z: " << posV[i].z << std::endl;
    //    std::cout << " x: " << vel1[i].x << " y: " << vel1[i].y << " z: " << vel1[i].z << std::endl;
    //    std::cout << " x: " << vel2[i].x << " y: " << vel2[i].y << " z: " << vel2[i].z << std::endl;
    //    std::cout << " --------------------------------- " << std::endl;
    //    
    //}

    // How to copy data to the GPU
    hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

    // Wrap device vectors in thrust iterators for use with thrust.
    thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
    thrust::device_ptr<int> dev_thrust_values(dev_intValues);
    // LOOK-2.1 Example for using thrust::sort_by_key
    thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

    // How to copy data back to the CPU side from the GPU
    hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
    hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
    checkCUDAErrorWithLine("memcpy back failed!");

    /*std::cout << "after unstable sort: " << std::endl;
    for (int i = 0; i < N; i++) {
        std::cout << "  key: " << intKeys[i];
        std::cout << " value: " << intValues[i] << std::endl;
    }*/

    // cleanup
    hipFree(dev_intKeys);
    hipFree(dev_intValues);
    checkCUDAErrorWithLine("hipFree failed!");
    return;
}
