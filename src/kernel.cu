#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

#include <>

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 250.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
glm::vec3* dev_pos2;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  // 27 grid
  gridCellWidth = 2.f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));

  hipMalloc((void**)&dev_pos2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos2 failed!");

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
  glm::vec3 perceivedCenter = glm::vec3(0.0f, 0.0f, 0.0f);
  glm::vec3 c = glm::vec3(0.0f, 0.0f, 0.0f);
  glm::vec3 perceivedVelocity = glm::vec3(0.0f, 0.0f, 0.0f);
  float neighborNumR1 = 0;
  float neighborNumR3 = 0;

  for (int i = 0; i < N; i++) {
    if (i == iSelf)
      continue;

    float distance = glm::distance(pos[i], pos[iSelf]);

    // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
    if (distance < rule1Distance) {
      perceivedCenter += pos[i];
      neighborNumR1++;
    }

    // Rule 2: boids try to stay a distance d away from each other
    if (distance < rule2Distance) {
      c -= (pos[i] - pos[iSelf]);
    }

    // Rule 3: boids try to match the speed of surrounding boids
    if (distance < rule3Distance) {
      perceivedVelocity += vel[i];
      neighborNumR3++;
    }
  }

  glm::vec3 dv = glm::vec3(0.0f, 0.0f, 0.0f);

  if (neighborNumR1 > 0) 
    dv += (perceivedCenter / neighborNumR1 - pos[iSelf]) * rule1Scale;

  dv += c* rule2Scale;

  if(neighborNumR3 > 0)
    dv += (perceivedVelocity / neighborNumR3) * rule3Scale;

  return dv;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }

  // Compute a new velocity based on pos and vel1
  glm::vec3 dv = computeVelocityChange(N, index, pos, vel1) + vel1[index];

  // Clamp the speed
  float speed = glm::length(dv);
  if (speed > maxSpeed) {
    dv = glm::normalize(dv) * maxSpeed;
  }

  // Record the new velocity into vel2. Question: why NOT vel1?
  vel2[index] = dv;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index >= N) {
    return;
  }

  // - Label each boid with the index of its grid cell.
  int iX = glm::floor((pos[index].x - gridMin.x) * inverseCellWidth);
  int iY = glm::floor((pos[index].y - gridMin.y) * inverseCellWidth);
  int iZ = glm::floor((pos[index].z - gridMin.z) * inverseCellWidth);
  int index1D = gridIndex3Dto1D(iX, iY, iZ, gridResolution);

  // - Set up a parallel array of integer indices as pointers to the actual
  //   boid data in pos and vel1/vel2
  indices[index] = index;
  gridIndices[index] = index1D;
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index >= N) {
    return;
  }

  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"
  int gridIndex = particleGridIndices[index];

  if (index == 0 || gridIndex != particleGridIndices[index - 1])
    gridCellStartIndices[gridIndex] = index;
  if (index == N - 1 || gridIndex != particleGridIndices[index + 1])
    gridCellEndIndices[gridIndex] = index;
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index >= N) {
    return;
  }

  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
  glm::vec3 gridCell = (pos[index] - gridMin) * inverseCellWidth;
  glm::vec3 gridCellIndex = glm::floor(gridCell);

  // - Identify which cells may contain neighbors. This isn't always 8.
  glm::vec3 gridCellFrac = gridCell - gridCellIndex;
  glm::vec3 startGridIndex, endGridIndex;
  startGridIndex.x = (gridCellIndex.x > 0 && gridCellFrac.x <= 0.5f) ? 1.f : 0.f;
  startGridIndex.y = (gridCellIndex.y > 0 && gridCellFrac.y <= 0.5f) ? 1.f : 0.f;
  startGridIndex.z = (gridCellIndex.z > 0 && gridCellFrac.z <= 0.5f) ? 1.f : 0.f;
  endGridIndex.x = (gridCellIndex.x < gridResolution - 1 && gridCellFrac.x > 0.5f) ? 1.f : 0.f;
  endGridIndex.y = (gridCellIndex.y < gridResolution - 1 && gridCellFrac.y > 0.5f) ? 1.f : 0.f;
  endGridIndex.z = (gridCellIndex.z < gridResolution - 1 && gridCellFrac.z > 0.5f) ? 1.f : 0.f;

  // - For each cell, read the start/end indices in the boid pointer array.
  glm::vec3 perceivedCenter = glm::vec3(0.0f, 0.0f, 0.0f);
  glm::vec3 c = glm::vec3(0.0f, 0.0f, 0.0f);
  glm::vec3 perceivedVelocity = glm::vec3(0.0f, 0.0f, 0.0f);

  int neighborsNumR1 = 0;
  int neighborsNumR3 = 0;
  for (int z = gridCellIndex.z - startGridIndex.z; z <= gridCellIndex.z + endGridIndex.z; z++) {
    for (int y = gridCellIndex.y - startGridIndex.y; y <= gridCellIndex.y + endGridIndex.y; y++) {
      for (int x = gridCellIndex.x - startGridIndex.x; x <= gridCellIndex.x + endGridIndex.x; x++) {
        int neighborGridIndex = gridIndex3Dto1D(x, y, z, gridResolution);

        if (gridCellStartIndices[neighborGridIndex] == -1) {
          continue;
        }

        // - Access each boid in the cell and compute velocity change from
        //   the boids rules, if this boid is within the neighborhood distance.
        int startIndex = gridCellStartIndices[neighborGridIndex];
        int endIndex = gridCellEndIndices[neighborGridIndex];
        for (int i = startIndex; i <= endIndex; i++) {
          int boidIndex = particleArrayIndices[i];
          if (boidIndex == index)
            continue;

          glm::vec3 pos1 = pos[index], pos2 = pos[boidIndex];
          float distance = glm::distance(pos1, pos2);
          if (distance < rule1Distance) {
            perceivedCenter += pos2;
            neighborsNumR1++;
          }
          if (distance < rule2Distance) {
            c -= (pos2 - pos1);
          }
          if (distance < rule3Distance) {
            perceivedVelocity += vel1[boidIndex];
            neighborsNumR3++;
          }

        }
      }
    }
  }

  glm::vec3 velocityChanged = glm::vec3(0.f, 0.f, 0.f);

  if (neighborsNumR1 > 0)
    velocityChanged += (perceivedCenter / (float)neighborsNumR1 - pos[index]) * rule1Scale;
  if (neighborsNumR3 > 0)
    velocityChanged += perceivedVelocity / (float)neighborsNumR3 * rule3Scale;
  velocityChanged += c * rule2Scale;

  // - Clamp the speed change before putting the new speed in vel2
  velocityChanged += vel1[index];
  if (glm::length(velocityChanged) > maxSpeed)
    velocityChanged = glm::normalize(velocityChanged) * maxSpeed;

  vel2[index] = velocityChanged;
}

__global__ void kernRearrangeData(
  int N, int* particleArrayIndices,
  glm::vec3* pos1, glm::vec3* pos2, glm::vec3* vel1, glm::vec3* vel2) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index >= N) {
    return;
  }

  int particleIndex = particleArrayIndices[index];
  pos2[index] = pos1[particleIndex];
  vel2[index] = vel1[particleIndex];
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index >= N) {
    return;
  }

  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
    // - Identify the grid cell that this particle is in
  glm::vec3 gridCell = (pos[index] - gridMin) * inverseCellWidth;
  glm::vec3 gridCellIndex = glm::floor(gridCell);

  // - Identify which cells may contain neighbors. This isn't always 8.
  glm::vec3 gridCellFrac = gridCell - gridCellIndex;
  glm::vec3 startGridIndex, endGridIndex;
  startGridIndex.x = (gridCellIndex.x > 0 && gridCellFrac.x <= 0.5f) ? 1.f : 0.f;
  startGridIndex.y = (gridCellIndex.y > 0 && gridCellFrac.y <= 0.5f) ? 1.f : 0.f;
  startGridIndex.z = (gridCellIndex.z > 0 && gridCellFrac.z <= 0.5f) ? 1.f : 0.f;
  endGridIndex.x = (gridCellIndex.x < gridResolution - 1 && gridCellFrac.x > 0.5f) ? 1.f : 0.f;
  endGridIndex.y = (gridCellIndex.y < gridResolution - 1 && gridCellFrac.y > 0.5f) ? 1.f : 0.f;
  endGridIndex.z = (gridCellIndex.z < gridResolution - 1 && gridCellFrac.z > 0.5f) ? 1.f : 0.f;

  /*
    startGridIndex.x = (gridCellIndex.x > 0) ? 1.f : 0.f;
  startGridIndex.y = (gridCellIndex.y > 0) ? 1.f : 0.f;
  startGridIndex.z = (gridCellIndex.z > 0) ? 1.f : 0.f;
  endGridIndex.x = (gridCellIndex.x < gridResolution - 1) ? 1.f : 0.f;
  endGridIndex.y = (gridCellIndex.y < gridResolution - 1) ? 1.f : 0.f;
  endGridIndex.z = (gridCellIndex.z < gridResolution - 1) ? 1.f : 0.f;
  */

  glm::vec3 perceivedCenter = glm::vec3(0.0f, 0.0f, 0.0f);
  glm::vec3 c = glm::vec3(0.0f, 0.0f, 0.0f);
  glm::vec3 perceivedVelocity = glm::vec3(0.0f, 0.0f, 0.0f);

  int neighborsNumR1 = 0;
  int neighborsNumR3 = 0;
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  for (int z = gridCellIndex.z - startGridIndex.z; z <= gridCellIndex.z + endGridIndex.z; z++) {
    for (int y = gridCellIndex.y - startGridIndex.y; y <= gridCellIndex.y + endGridIndex.y; y++) {
      for (int x = gridCellIndex.x - startGridIndex.x; x <= gridCellIndex.x + endGridIndex.x; x++) {
        int neighborGridIndex = gridIndex3Dto1D(x, y, z, gridResolution);

        if (gridCellStartIndices[neighborGridIndex] == -1) {
          continue;
        }
        // - For each cell, read the start/end indices in the boid pointer array.
        int startIndex = gridCellStartIndices[neighborGridIndex];
        int endIndex = gridCellEndIndices[neighborGridIndex];
        for (int boidIndex = startIndex; boidIndex <= endIndex; boidIndex++) {
          // int boidIndex = particleArrayIndices[i];
          if (boidIndex == index)
            continue;

          // - Access each boid in the cell and compute velocity change from
          //   the boids rules, if this boid is within the neighborhood distance.
          glm::vec3 pos1 = pos[index], pos2 = pos[boidIndex];
          float distance = glm::distance(pos1, pos2);
          if (distance < rule1Distance) {
            perceivedCenter += pos2;
            neighborsNumR1++;
          }
          if (distance < rule2Distance) {
            c -= (pos2 - pos1);
          }
          if (distance < rule3Distance) {
            perceivedVelocity += vel1[boidIndex];
            neighborsNumR3++;
          }
        }
      }
    }
  }
  
  glm::vec3 velocityChanged = glm::vec3(0.f, 0.f, 0.f);

  if (neighborsNumR1 > 0)
    velocityChanged += (perceivedCenter / (float)neighborsNumR1 - pos[index]) * rule1Scale;
  if (neighborsNumR3 > 0)
    velocityChanged += perceivedVelocity / (float)neighborsNumR3 * rule3Scale;
  velocityChanged += c * rule2Scale;

  // - Clamp the speed change before putting the new speed in vel2
  velocityChanged += vel1[index];
  if (glm::length(velocityChanged) > maxSpeed)
    velocityChanged = glm::normalize(velocityChanged) * maxSpeed;

  vel2[index] = velocityChanged;
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
  kernUpdateVelocityBruteForce << <fullBlocksPerGrid, blockSize >> > (
    numObjects, dev_pos, dev_vel1, dev_vel2);
  checkCUDAErrorWithLine("kernUpdateVelocityBruteForce failed!");

  kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt, dev_pos, dev_vel2);
  checkCUDAErrorWithLine("kernUpdatePos failed!");

  // TODO-1.2 ping-pong the velocity buffers
  hipMemcpy(dev_vel1, dev_vel2, numObjects * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed
  dim3 fullBlocksPerGridBoids((numObjects + blockSize - 1) / blockSize);
  dim3 fullBlocksPerGridCells((gridCellCount + blockSize - 1) / blockSize);

  kernComputeIndices << <fullBlocksPerGridBoids, blockSize >> > (
    numObjects, gridSideCount, gridMinimum, gridInverseCellWidth,
    dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

  dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);
  dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);
  thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

  kernResetIntBuffer << <fullBlocksPerGridCells, blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1);

  kernIdentifyCellStartEnd << <fullBlocksPerGridBoids, blockSize >> > (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

  kernUpdateVelNeighborSearchScattered << <fullBlocksPerGridBoids, blockSize >> > (
    numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth,
    dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);

  kernUpdatePos << <fullBlocksPerGridBoids, blockSize >> > (numObjects, dt, dev_pos, dev_vel2);

  hipMemcpy(dev_vel1, dev_vel2, numObjects * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  dim3 fullBlocksPerGridBoids((numObjects + blockSize - 1) / blockSize);
  dim3 fullBlocksPerGridCells((gridCellCount + blockSize - 1) / blockSize);

  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  kernComputeIndices << <fullBlocksPerGridBoids, blockSize >> > (
    numObjects, gridSideCount, gridMinimum, gridInverseCellWidth,
    dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);
  dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);
  thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  kernResetIntBuffer << <fullBlocksPerGridCells, blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1);
  kernIdentifyCellStartEnd << <fullBlocksPerGridBoids, blockSize >> > (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // pos1 -> pos2 and vel1 -> vel2
  kernRearrangeData << <fullBlocksPerGridBoids, blockSize >> > (numObjects, dev_particleArrayIndices, dev_pos, dev_pos2, dev_vel1, dev_vel2);

  // - Perform velocity updates using neighbor search (pos2 and vel2 -> vel1)
  kernUpdateVelNeighborSearchCoherent << <fullBlocksPerGridBoids, blockSize >> > (
    numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth,
    dev_gridCellStartIndices, dev_gridCellEndIndices, dev_pos2, dev_vel2, dev_vel1);

  // - Update positions
  kernUpdatePos << <fullBlocksPerGridBoids, blockSize >> > (numObjects, dt, dev_pos2, dev_vel1);

  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
  hipMemcpy(dev_pos, dev_pos2, numObjects * sizeof(glm::vec3), hipMemcpyDeviceToDevice);
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);

  hipFree(dev_pos2);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
