#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"
#include <>

#define _1X_WIDTH_GRID_ 0

// potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3* dev_pos2; // coherent buffer when aligning dev_pos with grid
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// Grid parameters based on simulation parameters.
// These are automatically computed in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* basic helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");
  hipMalloc((void**)&dev_pos2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos2 failed!");
  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");
  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects, dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // computing grid params
  #if _1X_WIDTH_GRID_
  gridCellWidth = std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  #else 
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  #endif
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  hipMalloc((void**)&dev_particleArrayIndices, numObjects * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");
  hipMalloc((void**)&dev_particleGridIndices, numObjects * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");
  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");
  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);
  dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3* pos, float* vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3* vel, float* vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float* vbodptr_positions, float* vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3* pos, const glm::vec3* vel) {
  // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
  // Rule 2: boids try to stay a distance d away from each other
  // Rule 3: boids try to match the speed of surrounding boids
  glm::vec3 perceivedCenter = glm::vec3(0.f);
  glm::vec3 c = glm::vec3(0.f);
  glm::vec3 perceivedVel = glm::vec3(0.f);
  int neighborInRule1Dist = 0;
  int neighborInRule3Dist = 0;
  for (int i = 0; i < N; ++i) {
    if (i != iSelf) {
      float dist = glm::distance(pos[i], pos[iSelf]);
      if (dist < rule1Distance) {
        perceivedCenter += pos[i];
        ++neighborInRule1Dist;
      }
      if (dist < rule2Distance) {
        c -= (pos[i] - pos[iSelf]);
      }
      if (dist < rule3Distance) {
        perceivedVel += vel[i];
        ++neighborInRule3Dist;
      }
    }
  }

  glm::vec3 velChange = glm::vec3(0.f);
  if (neighborInRule1Dist > 0) {
    perceivedCenter /= neighborInRule1Dist;
    velChange += (perceivedCenter - pos[iSelf]) * rule1Scale;
  }
  velChange += c * rule2Scale;
  if (neighborInRule3Dist > 0) {
    perceivedVel /= neighborInRule3Dist;
    velChange += perceivedVel * rule3Scale;
  }
  return velChange;
}

/**
* implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (i >= N) { return; }
  // Compute a new velocity based on pos and vel1
  // Record in new vel2. (since vel1 may still need to be referenced for rule3)
  vel2[i] = vel1[i] + computeVelocityChange(N, i, pos, vel1);

  // Clamp speed if needed
  if (glm::length(vel2[i]) > maxSpeed) {
    vel2[i] = glm::normalize(vel2[i]) * maxSpeed;
  }
}

/**
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, glm::vec3* pos, int* indices, int* gridIndices) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (i >= N) { return; }
  // parallel array of indices as pointers to the actual boid data in pos and vel1/vel2
  indices[i] = i;
  // Label each boid with the index of its grid cell.
  glm::vec3 posInCell = (pos[i] - gridMin) * inverseCellWidth;
  gridIndices[i] = gridIndex3Dto1D(int(posInCell.x), int(posInCell.y), int(posInCell.z),
    gridResolution);
}

// indicating that a cell does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (i >= N) { return; }
  // Identify the start point of each cell in the gridIndices array.
  // parallel unrolling of a loop
  int currBoidIdx = particleGridIndices[i]; // index IN dev_pos or whatever
  if (i == N - 1) {
    gridCellEndIndices[currBoidIdx] = i;
    return;
  }

  int nextBoidIdx = particleGridIndices[i + 1];
  if (i == 0) {
    gridCellStartIndices[currBoidIdx] = i;
  }
  // "this index doesn't match the one before it, must be a new cell!"
  if (currBoidIdx != nextBoidIdx) {
    gridCellEndIndices[currBoidIdx] = i + 1;
    gridCellStartIndices[nextBoidIdx] = i + 1;
  } 
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // Update a boid's velocity using the uniform grid
  int currIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (currIdx >= N) { return; }
  // Identify the grid cell that this particle is in
  glm::vec3 posRelativeToGrid = pos[currIdx] - gridMin;
  glm::vec3 posInCell = posRelativeToGrid * inverseCellWidth;
  glm::vec3 posOfCell = glm::vec3(int(posInCell.x), int(posInCell.y), int(posInCell.z));
  int gridCellIdx = gridIndex3Dto1D(posOfCell.x, posOfCell.y, posOfCell.z, gridResolution);
  // Identify which cells may contain neighbors. This isn't always 8.
  glm::vec3 perceivedCenter = glm::vec3(0.f);
  glm::vec3 c = glm::vec3(0.f);
  glm::vec3 perceivedVel = glm::vec3(0.f);
  int neighborInRule1Dist = 0;
  int neighborInRule3Dist = 0;

  #if _1X_WIDTH_GRID_ // 27 neighbors
    int xStart = -1; int xEnd = 1;
    int yStart = -1; int yEnd = 1;
    int zStart = -1; int zEnd = 1;
  #else // 8 neighbors
    int xStart = 0; int xEnd = 0;
    int yStart = 0; int yEnd = 0;
    int zStart = 0; int zEnd = 0;
    glm::vec3 posRelativeToCell = posRelativeToGrid - (posOfCell * cellWidth);
    if (posRelativeToCell.x < cellWidth / 2) { xStart = -1; } else { xEnd = 1; }
    if (posRelativeToCell.y < cellWidth / 2) { yStart = -1; } else { yEnd = 1; }
    if (posRelativeToCell.z < cellWidth / 2) { zStart = -1; } else { zEnd = 1; }
  #endif
  //For each cell, read the start/end indices in the boid pointer array.
   for (int x = xStart; x <= xEnd; ++x) {
    for (int y = yStart; y <= yEnd; ++y) {
      for (int z = zStart; z <= zEnd; ++z) {
        int neighborCellIdx = gridCellIdx + gridIndex3Dto1D(x, y, z, gridResolution);
        // gridRes ^ 3 == gridCellCount
        if (neighborCellIdx < 0 || 
          gridResolution * gridResolution * gridResolution <= neighborCellIdx) { continue; }
        int gridStart = gridCellStartIndices[neighborCellIdx];
        if (gridStart == -1) { continue; } // as set from ResetIntBuffer

        // Access each boid in the cell, compute vel change from boids rules
        for (int arrIdx = gridStart; arrIdx < gridCellEndIndices[neighborCellIdx]; ++arrIdx) {
          int neighborIdx = particleArrayIndices[arrIdx];
          if (currIdx == neighborIdx) { continue; }
          // Rule 1: boids fly towards their local perceived center of mass
          // Rule 2: boids try to stay a distance d away from each other
          // Rule 3: boids try to match the speed of surrounding boids
          float dist = glm::distance(pos[neighborIdx], pos[currIdx]);
          if (dist < rule1Distance) {
            perceivedCenter += pos[neighborIdx];
            ++neighborInRule1Dist;
          }
          if (dist < rule2Distance) {
            c -= (pos[neighborIdx] - pos[currIdx]);
          }
          if (dist < rule3Distance) {
            perceivedVel += vel1[neighborIdx];
            ++neighborInRule3Dist;
          }
        }
      }
    }
  }
  glm::vec3 velChange = glm::vec3(0.f);
  if (neighborInRule1Dist > 0) {
    perceivedCenter /= neighborInRule1Dist;
    velChange += (perceivedCenter - pos[currIdx]) * rule1Scale;
  }
  velChange += c * rule2Scale;
  if (neighborInRule3Dist > 0) {
    perceivedVel /= neighborInRule3Dist;
    velChange += perceivedVel * rule3Scale;
  }
  vel2[currIdx] = vel1[currIdx] + velChange;
  // Clamp speed if needed
  if (glm::length(vel2[currIdx]) > maxSpeed) {
    vel2[currIdx] = glm::normalize(vel2[currIdx]) * maxSpeed;
  }
}

// maybe a better name is kern "align" bufs or "make coherent"
__global__ void kernShuffleBufs(int N, int *particleArrayIndices,
  glm::vec3* pos1, glm::vec3* pos2, glm::vec3* vel1, glm::vec3* vel2) {
  int currIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (currIdx >= N) { return; }
  pos2[currIdx] = pos1[particleArrayIndices[currIdx]];
  vel2[currIdx] = vel1[particleArrayIndices[currIdx]]; //2s become coheerent
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // gridCellStartIndices and gridCellEndIndices refer directly to pos and vel1.
  int currIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (currIdx >= N) { return; }
  // Identify the grid cell that this particle is in
  glm::vec3 posRelativeToGrid = pos[currIdx] - gridMin;
  glm::vec3 posInCell = posRelativeToGrid * inverseCellWidth;
  glm::vec3 posOfCell = glm::vec3(int(posInCell.x), int(posInCell.y), int(posInCell.z));
  int gridCellIdx = gridIndex3Dto1D(posOfCell.x, posOfCell.y, posOfCell.z, gridResolution);
  // Identify which cells may contain neighbors. This isn't always 8.
  glm::vec3 perceivedCenter = glm::vec3(0.f);
  glm::vec3 c = glm::vec3(0.f);
  glm::vec3 perceivedVel = glm::vec3(0.f);
  int neighborInRule1Dist = 0;
  int neighborInRule3Dist = 0;

  int xStart = 0; int xEnd = 0;
  int yStart = 0; int yEnd = 0;
  int zStart = 0; int zEnd = 0;
  glm::vec3 posRelativeToCell = posRelativeToGrid - (posOfCell * cellWidth);
  if (posRelativeToCell.x < cellWidth / 2) { xStart = -1; } else { xEnd = 1; }
  if (posRelativeToCell.y < cellWidth / 2) { yStart = -1; } else { yEnd = 1; }
  if (posRelativeToCell.z < cellWidth / 2) { zStart = -1; } else { zEnd = 1; }
  // For each cell, read the start/end indices in the boid pointer array.
  // DIFFERENCE: For best results, consider what order the cells should be
  // checked in to maximize the memory benefits of reordering the boids data.
  for (int x = xStart; x <= xEnd; ++x) {
    for (int y = yStart; y <= yEnd; ++y) {
      for (int z = zStart; z <= zEnd; ++z) {
        // gridRes ^ 3 == gridCellCount
        int neighborCellIdx = gridCellIdx + gridIndex3Dto1D(x, y, z, gridResolution);
        if (neighborCellIdx < 0 ||
          gridResolution * gridResolution * gridResolution <= neighborCellIdx) {
          continue;
        }
        int gridStart = gridCellStartIndices[neighborCellIdx];
        if (gridStart == -1) { continue; } // as set from ResetIntBuffer

        // Access each boid in the cell, compute vel change from boids rules
        for (int neighborIdx = gridStart; neighborIdx < gridCellEndIndices[neighborCellIdx]; ++neighborIdx) {
          if (currIdx == neighborIdx) { continue; }
          // Rule 1: boids fly towards their local perceived center of mass
          // Rule 2: boids try to stay a distance d away from each other
          // Rule 3: boids try to match the speed of surrounding boids
          float dist = glm::distance(pos[neighborIdx], pos[currIdx]);
          if (dist < rule1Distance) {
            perceivedCenter += pos[neighborIdx];
            ++neighborInRule1Dist;
          }
          if (dist < rule2Distance) {
            c -= (pos[neighborIdx] - pos[currIdx]);
          }
          if (dist < rule3Distance) {
            perceivedVel += vel1[neighborIdx];
            ++neighborInRule3Dist;
          }
        }
      }
    }
  }
  glm::vec3 velChange = glm::vec3(0.f);
  if (neighborInRule1Dist > 0) {
    perceivedCenter /= neighborInRule1Dist;
    velChange += (perceivedCenter - pos[currIdx]) * rule1Scale;
  }
  velChange += c * rule2Scale;
  if (neighborInRule3Dist > 0) {
    perceivedVel /= neighborInRule3Dist;
    velChange += perceivedVel * rule3Scale;
  }
  vel2[currIdx] = vel1[currIdx] + velChange;
  // Clamp speed if needed.. TODO: in this coherent STEP it goes to MAX SPEED almost
  // instantly. compare with non-coherent which takes a second-ish to go wild. this is 
  // almost certainly a symptom of t he problem.
  if (glm::length(vel2[currIdx]) > maxSpeed) {
    vel2[currIdx] = glm::normalize(vel2[currIdx]) * maxSpeed;
  }
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // use kernels to step the simulation forward in time.
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
  kernUpdateVelocityBruteForce<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, dev_vel1, dev_vel2);
  checkCUDAErrorWithLine("kernUpdateVelocityBruteForce failed!");
  // ping-pong velocity buffers
  std::swap(dev_vel1, dev_vel2);
  kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel1);
  checkCUDAErrorWithLine("kernUpdatePos failed!");
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // Uniform Grid Neighbor search using Thrust sort.
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
  // label each particle with its array index as well as its grid index.
  kernComputeIndices<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount,
    gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices,
    dev_particleGridIndices);
  checkCUDAErrorWithLine("kernComputeIndices failed!");
  // Unstable key sort using Thrust. stable sort isn't necessary
  thrust::sort_by_key(dev_thrust_particleGridIndices,
    dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

  // Naively unroll the loop for finding the start and end indices of each
  // cell's data pointers in the array of boid indices
  dim3 fullBlocksPerCell((gridCellCount + blockSize - 1) / blockSize);
  kernResetIntBuffer<<<fullBlocksPerCell, blockSize>>>(gridCellCount,
    dev_gridCellStartIndices, -1);
  checkCUDAErrorWithLine("kernResetIntBuffer failed!");
  kernIdentifyCellStartEnd<<<fullBlocksPerGrid, blockSize>>>(numObjects,
    dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
  checkCUDAErrorWithLine("kernIdentifyCellStartEnd failed!");
  // velocity updates using neighbor search
  kernUpdateVelNeighborSearchScattered<<<fullBlocksPerGrid, blockSize>>>(numObjects,
    gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth,
    dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices,
    dev_pos, dev_vel1, dev_vel2);
  checkCUDAErrorWithLine("kernUpdateVelNeighborSearchScattered failed!");
  // ping poing, Update positions
  std::swap(dev_vel1, dev_vel2);
  kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel1);
  checkCUDAErrorWithLine("kernUpdatePos failed!");
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // Uniform Grid Neighbor search w/ Thrust sort on cell-coherent data.
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
  // Label each particle with its array index as well as its grid index.
  kernComputeIndices<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount,
    gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices,
    dev_particleGridIndices);
  checkCUDAErrorWithLine("kernComputeIndices failed!");
  // Unstable key sort using Thrust. A stable sort isn't necessary
  thrust::sort_by_key(dev_thrust_particleGridIndices,
    dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);
  // Naively unroll the loop for finding the start and end indices of each
  // cell's data pointers in the array of boid indices
  dim3 fullBlocksPerCell((gridCellCount + blockSize - 1) / blockSize);
  kernResetIntBuffer<<<fullBlocksPerCell, blockSize>>>(gridCellCount,
    dev_gridCellStartIndices, -1);
  checkCUDAErrorWithLine("kernResetIntBuffer failed!");
  kernIdentifyCellStartEnd<<<fullBlocksPerGrid, blockSize>>>(numObjects,
    dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
  checkCUDAErrorWithLine("kernIdentifyCellStartEnd failed!");
  // BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  // the particle data in the simulation array.
  // CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  kernShuffleBufs<<<fullBlocksPerGrid, blockSize>>>(numObjects,
    dev_particleArrayIndices, dev_pos, dev_pos2, dev_vel1, dev_vel2);
  checkCUDAErrorWithLine("kernShuffleBufs failed!");
  // Perform velocity updates using neighbor search
  kernUpdateVelNeighborSearchCoherent<<<fullBlocksPerGrid, blockSize>>>(numObjects,
    gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth,
    dev_gridCellStartIndices, dev_gridCellEndIndices, dev_pos2,
    dev_vel2, dev_vel1); // swapped since vel2 is coherent, use vel1 as tmp
  checkCUDAErrorWithLine("kernUpdateVelNeighborSearchCoherent failed!");
  // Update positions Ping-pong buffers as needed
  std::swap(dev_pos, dev_pos2);
  // dev_vel1 should be coherent. not necesary to ping pong.
  kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects,
    dt, dev_pos, dev_vel1);
  checkCUDAErrorWithLine("kernUpdatePos failed!");
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);
  hipFree(dev_pos2);

  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);
}

void Boids::unitTest() {
  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
