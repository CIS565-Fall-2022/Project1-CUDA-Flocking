#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <cfloat>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"


#define SINGLE_WIDTH 1;

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
thrust::device_ptr<glm::vec3> dev_thrust_posArrayIndices;
thrust::device_ptr<glm::vec3> dev_thrust_velArrayIndices;
int* dev_gridIndicesCopy1;
int* dev_gridIndicesCopy2;
thrust::device_ptr<int> dev_thrust_gridIndicesCopy1;
thrust::device_ptr<int> dev_thrust_gridIndicesCopy2;


// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
    #if SINGLE_WIDTH
    gridCellWidth = std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
    #else
    gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
    #endif
    int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
    gridSideCount = 2 * halfSideCount;

    gridCellCount = gridSideCount * gridSideCount * gridSideCount;
    gridInverseCellWidth = 1.0f / gridCellWidth;
    float halfGridWidth = gridCellWidth * halfSideCount;
    gridMinimum.x -= halfGridWidth;
    gridMinimum.y -= halfGridWidth;
    gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
    hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
    checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");
    hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
    checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");
    
    hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
    checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");
    hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
    checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

    hipMalloc((void**)&dev_gridIndicesCopy1, N * sizeof(int));
    checkCUDAErrorWithLine("hipMalloc dev_gridIndicesCopy1 failed!");
    hipMalloc((void**)&dev_gridIndicesCopy2, N * sizeof(int));
    checkCUDAErrorWithLine("hipMalloc dev_gridIndicesCopy2 failed!");


  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
    glm::vec3 v1 = glm::vec3(0.0f);
    glm::vec3 v2 = glm::vec3(0.0f);
    glm::vec3 v3 = glm::vec3(0.0f);
    int num_neighbor1 = 0;
    int num_neighbor3 = 0;
    
    // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
    glm::vec3 perceived_center = glm::vec3(0.0f);
    // Rule 2: boids try to stay a distance d away from each other
    glm::vec3 c = glm::vec3(0.0f);
    // Rule 3: boids try to match the speed of surrounding boids
    glm::vec3 perceived_velocity = glm::vec3(0.0f);

    for (int i = 0; i < N; i++)
    {
        if (i != iSelf )
        {
            if (glm::length(pos[iSelf] - pos[i]) < rule1Distance)
            {
                perceived_center += pos[i];
                num_neighbor1++;
            }

            if (glm::length(pos[iSelf] - pos[i]) < rule2Distance)
            {
                c -= pos[i] - pos[iSelf];
            }
            if (glm::length(pos[iSelf] - pos[i]) < rule3Distance)
            {
                perceived_velocity += vel[i];
                num_neighbor3++;
            }
        }
    }
    if (num_neighbor1 > 0)
    {
        perceived_center /= num_neighbor1;
        v1 = (perceived_center - pos[iSelf]) * rule1Scale;
    }
    v2 = c * rule2Scale;
    if (num_neighbor3 > 0)
    {
        perceived_velocity /= num_neighbor3;
        v3 = perceived_velocity * rule3Scale;
    }
    return v1 + v2 + v3;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  // Compute a new velocity based on pos and vel1
  // Clamp the speed
  // Record the new velocity into vel2. Question: why NOT vel1?
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) {
        return;
    }

    glm::vec3 change = computeVelocityChange(N, index, pos, vel1);
    change += vel1[index];
    float speed = glm::length(change);
    if (speed > maxSpeed)
    {
        change *= maxSpeed / speed;
    }
    vel2[index] = change;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) { return; }
    indices[index] = index;

    glm::vec3 position = pos[index] - gridMin;
    glm::vec3 gridVec = floor(inverseCellWidth * position);
    gridIndices[index] = gridIndex3Dto1D(int(gridVec.x), int(gridVec.y), int(gridVec.z), gridResolution);
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= (N - 1)) { return; }

    if (particleGridIndices[index] != particleGridIndices[index + 1])
    {
        gridCellEndIndices[particleGridIndices[index]] = index;
        gridCellStartIndices[particleGridIndices[index+1]] = index+1;
    }
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth, int gridCellCount,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) { return; }

    glm::vec3 v1 = glm::vec3(0.0f);
    glm::vec3 v2 = glm::vec3(0.0f);
    glm::vec3 v3 = glm::vec3(0.0f);
    int num_neighbor1 = 0;
    int num_neighbor3 = 0;

    // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
    glm::vec3 perceived_center = glm::vec3(0.0f);
    // Rule 2: boids try to stay a distance d away from each other
    glm::vec3 c = glm::vec3(0.0f);
    // Rule 3: boids try to match the speed of surrounding boids
    glm::vec3 perceived_velocity = glm::vec3(0.0f);


    glm::vec3 thisGridPos = pos[index] - gridMin;
    glm::vec3 thisCellVec = floor(inverseCellWidth * thisGridPos);

    int thisCellIndex = gridIndex3Dto1D(int(thisCellVec.x), int(thisCellVec.y), int(thisCellVec.z), gridResolution);

#if SINGLE_WIDTH
    for (int i = -1; i < 2; i++)
    {
        for (int j = -1; j < 2; j++)
        {
            for (int k = -1; k < 2; k++)
            {
                glm::vec3 neighborCellVec = thisCellVec + glm::vec3(i, j, k);
                int neighborCellNum = gridIndex3Dto1D(int(neighborCellVec.x), int(neighborCellVec.y),
                    int(neighborCellVec.z), gridResolution);
                if (neighborCellNum < 0 || neighborCellNum >= gridCellCount)
                {
                    //out of boundary
                    continue;
                }
                int thisNeighborStartIndex = gridCellStartIndices[neighborCellNum];
                if (thisNeighborStartIndex == -1)
                {
                    //this grid cell doesn't contain any boids, just skip it
                    continue;
                }
                int thisNeighborEndIndex = gridCellEndIndices[neighborCellNum];

                for (int w = thisNeighborStartIndex; w <= thisNeighborEndIndex; w++)
                {
                    int neighborBoidIndex = particleArrayIndices[w];
                    if (neighborBoidIndex != index)
                    {
                        if (glm::length(pos[index] - pos[neighborBoidIndex]) < rule1Distance)
                        {
                            perceived_center += pos[neighborBoidIndex];
                            num_neighbor1++;
                        }

                        if (glm::length(pos[index] - pos[neighborBoidIndex]) < rule2Distance)
                        {
                            c -= pos[neighborBoidIndex] - pos[index];
                        }
                        if (glm::length(pos[index] - pos[neighborBoidIndex]) < rule3Distance)
                        {
                            perceived_velocity += vel1[neighborBoidIndex];
                            num_neighbor3++;
                        }
                    }
                }
            }
        }
    }
#else
    glm::vec3 thisCellMin = thisCellVec * cellWidth; // the position of the minimum point of this cell
    float halfWidth = cellWidth / 2;
    float xDifference = thisGridPos.x - thisCellMin.x - halfWidth;
    float yDifference = thisGridPos.y - thisCellMin.y - halfWidth;
    float zDifference = thisGridPos.z - thisCellMin.z - halfWidth;
    int xBias = 0;
    int yBias = 0;
    int zBias = 0;
    if (fabs(xDifference) > FLT_EPSILON)
    {
        xBias = xDifference > 0 ? 1 : -1;
    }
    if (fabs(yDifference) > FLT_EPSILON)
    {
        yBias = yDifference > 0 ? 1 : -1;
    }
    if (fabs(zDifference) > FLT_EPSILON)
    {
        zBias = zDifference > 0 ? 1 : -1;
    }
    
    for (int i = 0; i < 2; i++)
    {
        for (int j = 0; j < 2; j++)
        {
            for (int k = 0; k < 2; k++)
            {
                // test for if the particle is on the middle lines
                // if one of the conditions is true, we only test for 4 cells
                // if two are true, we test for 2 cells
                // if three are true, then the particle is at the center of the cell, we only test for this 1 cell
                if ((xBias == 0 && i == 1)|| (yBias == 0 && j == 1)|| (zBias == 0 && k == 1))
                {
                    continue;
                }
                glm::vec3 neighborCellVec = thisCellVec + glm::vec3(i * xBias, j * yBias, k * zBias);
                int neighborCellNum = gridIndex3Dto1D(int(neighborCellVec.x), int(neighborCellVec.y),
                    int(neighborCellVec.z), gridResolution);
                if (neighborCellNum < 0 || neighborCellNum >= gridCellCount)
                {
                    //out of boundary
                    continue;
                }
                int thisNeighborStartIndex = gridCellStartIndices[neighborCellNum];
                if (thisNeighborStartIndex == -1)
                {
                    //this grid cell doesn't contain any boids, just skip it
                    continue;
                }
                int thisNeighborEndIndex = gridCellEndIndices[neighborCellNum];

                for (int w = thisNeighborStartIndex; w <= thisNeighborEndIndex; w++)
                {
                    int neighborBoidIndex = particleArrayIndices[w];
                    if (neighborBoidIndex != index)
                    {
                        if (glm::length(pos[index] - pos[neighborBoidIndex]) < rule1Distance)
                        {
                            perceived_center += pos[neighborBoidIndex];
                            num_neighbor1++;
                        }

                        if (glm::length(pos[index] - pos[neighborBoidIndex]) < rule2Distance)
                        {
                            c -= pos[neighborBoidIndex] - pos[index];
                        }
                        if (glm::length(pos[index] - pos[neighborBoidIndex]) < rule3Distance)
                        {
                            perceived_velocity += vel1[neighborBoidIndex];
                            num_neighbor3++;
                        }
                    }
                }
            }
        }
    }
#endif
    if (num_neighbor1 > 0)
    {
        perceived_center /= num_neighbor1;
        v1 = (perceived_center - pos[index]) * rule1Scale;
    }
    v2 = c * rule2Scale;
    if (num_neighbor3 > 0)
    {
        perceived_velocity /= num_neighbor3;
        v3 = perceived_velocity * rule3Scale;
    }
    glm::vec3 change =  v1 + v2 + v3;

    //Clamp the speed and store it into vel2
    change += vel1[index];
    float speed = glm::length(change);
    if (speed > maxSpeed)
    {
        change *= maxSpeed / speed;
    }
    vel2[index] = change;
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth, int gridCellCount,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2

    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) { return; }

    glm::vec3 v1 = glm::vec3(0.0f);
    glm::vec3 v2 = glm::vec3(0.0f);
    glm::vec3 v3 = glm::vec3(0.0f);
    int num_neighbor1 = 0;
    int num_neighbor3 = 0;

    // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
    glm::vec3 perceived_center = glm::vec3(0.0f);
    // Rule 2: boids try to stay a distance d away from each other
    glm::vec3 c = glm::vec3(0.0f);
    // Rule 3: boids try to match the speed of surrounding boids
    glm::vec3 perceived_velocity = glm::vec3(0.0f);


    glm::vec3 thisGridPos = pos[index] - gridMin;
    glm::vec3 thisCellVec = floor(inverseCellWidth * thisGridPos);

    int thisCellIndex = gridIndex3Dto1D(int(thisCellVec.x), int(thisCellVec.y), int(thisCellVec.z), gridResolution);

#if SINGLE_WIDTH
    for (int k = -1; k < 2; k++)
    {
        for (int j = -1; j < 2; j++)
        {
            for (int i = -1; i < 2; i++)
            {
                glm::vec3 neighborCellVec = thisCellVec + glm::vec3(i, j, k);
                int neighborCellNum = gridIndex3Dto1D(int(neighborCellVec.x), int(neighborCellVec.y),
                    int(neighborCellVec.z), gridResolution);
                if (neighborCellNum < 0 || neighborCellNum >= gridCellCount)
                {
                    //out of boundary
                    continue;
                }
                int thisNeighborStartIndex = gridCellStartIndices[neighborCellNum];
                if (thisNeighborStartIndex == -1)
                {
                    //this grid cell doesn't contain any boids, just skip it
                    continue;
                }
                int thisNeighborEndIndex = gridCellEndIndices[neighborCellNum];

                for (int w = thisNeighborStartIndex; w <= thisNeighborEndIndex; w++)
                {
                    int neighborBoidIndex = w;
                    if (neighborBoidIndex != index)
                    {
                        if (glm::length(pos[index] - pos[neighborBoidIndex]) < rule1Distance)
                        {
                            perceived_center += pos[neighborBoidIndex];
                            num_neighbor1++;
                        }

                        if (glm::length(pos[index] - pos[neighborBoidIndex]) < rule2Distance)
                        {
                            c -= pos[neighborBoidIndex] - pos[index];
                        }
                        if (glm::length(pos[index] - pos[neighborBoidIndex]) < rule3Distance)
                        {
                            perceived_velocity += vel1[neighborBoidIndex];
                            num_neighbor3++;
                        }
                    }
                }
            }
        }
    }
#else
    glm::vec3 thisCellMin = thisCellVec * cellWidth; // the position of the minimum point of this cell
    float halfWidth = cellWidth / 2;
    float xDifference = thisGridPos.x - thisCellMin.x - halfWidth;
    float yDifference = thisGridPos.y - thisCellMin.y - halfWidth;
    float zDifference = thisGridPos.z - thisCellMin.z - halfWidth;
    int xBias = 0;
    int yBias = 0;
    int zBias = 0;
    if (fabs(xDifference) > FLT_EPSILON)
    {
        xBias = xDifference > 0 ? 1 : -1;
    }
    if (fabs(yDifference) > FLT_EPSILON)
    {
        yBias = yDifference > 0 ? 1 : -1;
    }
    if (fabs(zDifference) > FLT_EPSILON)
    {
        zBias = zDifference > 0 ? 1 : -1;
    }

    for (int k = 0; k < 2; k++)
    {
        for (int j = 0; j < 2; j++)
        {
            for (int i = 0; i < 2; i++)
            {
                // test for if the particle is on the middle lines
                // if one of the conditions is true, we only test for 4 cells
                // if two are true, we test for 2 cells
                // if three are true, then the particle is at the center of the cell, we only test for this 1 cell
                if ((xBias == 0 && i == 1) || (yBias == 0 && j == 1) || (zBias == 0 && k == 1))
                {
                    continue;
                }
                glm::vec3 neighborCellVec = thisCellVec + glm::vec3(i * xBias, j * yBias, k * zBias);
                int neighborCellNum = gridIndex3Dto1D(int(neighborCellVec.x), int(neighborCellVec.y),
                    int(neighborCellVec.z), gridResolution);
                if (neighborCellNum < 0 || neighborCellNum >= gridCellCount)
                {
                    //out of boundary
                    continue;
                }
                int thisNeighborStartIndex = gridCellStartIndices[neighborCellNum];
                if (thisNeighborStartIndex == -1)
                {
                    //this grid cell doesn't contain any boids, just skip it
                    continue;
                }
                int thisNeighborEndIndex = gridCellEndIndices[neighborCellNum];

                for (int w = thisNeighborStartIndex; w <= thisNeighborEndIndex; w++)
                {
                    int neighborBoidIndex = w;
                    if (neighborBoidIndex != index)
                    {
                        if (glm::length(pos[index] - pos[neighborBoidIndex]) < rule1Distance)
                        {
                            perceived_center += pos[neighborBoidIndex];
                            num_neighbor1++;
                        }

                        if (glm::length(pos[index] - pos[neighborBoidIndex]) < rule2Distance)
                        {
                            c -= pos[neighborBoidIndex] - pos[index];
                        }
                        if (glm::length(pos[index] - pos[neighborBoidIndex]) < rule3Distance)
                        {
                            perceived_velocity += vel1[neighborBoidIndex];
                            num_neighbor3++;
                        }
                    }
                }
            }
        }
    }
#endif
    if (num_neighbor1 > 0)
    {
        perceived_center /= num_neighbor1;
        v1 = (perceived_center - pos[index]) * rule1Scale;
    }
    v2 = c * rule2Scale;
    if (num_neighbor3 > 0)
    {
        perceived_velocity /= num_neighbor3;
        v3 = perceived_velocity * rule3Scale;
    }
    glm::vec3 change = v1 + v2 + v3;

    //Clamp the speed and store it into vel2
    change += vel1[index];
    float speed = glm::length(change);
    if (speed > maxSpeed)
    {
        change *= maxSpeed / speed;
    }
    vel2[index] = change;
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
    // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
    kernUpdateVelocityBruteForce <<<fullBlocksPerGrid, blockSize >>> (numObjects, dev_pos, dev_vel1, dev_vel2);
    checkCUDAErrorWithLine("kernUpdateVelocityBruteForce failed!");
    kernUpdatePos <<<fullBlocksPerGrid, blockSize >>> (numObjects, dt, dev_pos, dev_vel2);
    checkCUDAErrorWithLine("kernUpdatePos failed!");
    // TODO-1.2 ping-pong the velocity buffers
    glm::vec3* temp = dev_vel1;
    dev_vel1 = dev_vel2;
    dev_vel2 = temp;
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed
    
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize); // block number for boid buffer
    dim3 fullBlocksForCell((gridCellCount + blockSize - 1) / blockSize);//block number for grid cell buffer

    kernComputeIndices <<<fullBlocksPerGrid, blockSize >>> (numObjects, gridSideCount,
        gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
    checkCUDAErrorWithLine("kernComputeIndices failed!");

    // Wrap device vectors in thrust iterators for use with thrust.
    dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);
    dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);
    //thrust::sort_by_key
    thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

    kernResetIntBuffer<<<fullBlocksForCell, blockSize >>> (gridCellCount, dev_gridCellStartIndices, -1);
    checkCUDAErrorWithLine("kernResetIntBuffer failed!");

    kernIdentifyCellStartEnd<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
    checkCUDAErrorWithLine("kernIdentifyCellStartEnd failed!");

    kernUpdateVelNeighborSearchScattered<<<fullBlocksPerGrid, blockSize >>>(numObjects, gridSideCount, gridMinimum,
        gridInverseCellWidth, gridCellWidth, gridCellCount, dev_gridCellStartIndices, dev_gridCellEndIndices,
        dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);
    checkCUDAErrorWithLine("kernUpdateVelNeighborSearchScattered failed!");

    kernUpdatePos <<<fullBlocksPerGrid, blockSize >>> (numObjects, dt, dev_pos, dev_vel2);
    checkCUDAErrorWithLine("kernUpdatePos failed!");

    glm::vec3* temp = dev_vel1;
    dev_vel1 = dev_vel2;
    dev_vel2 = temp;
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.


    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize); // block number for boid buffer
    dim3 fullBlocksForCell((gridCellCount + blockSize - 1) / blockSize);//block number for grid cell buffer

    kernComputeIndices <<<fullBlocksPerGrid, blockSize >>> (numObjects, gridSideCount,
        gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
    checkCUDAErrorWithLine("kernComputeIndices failed!");


    //additional index buffer for sorting pos and vel1
    hipMemcpy(dev_gridIndicesCopy1, dev_particleGridIndices, numObjects * sizeof(int), hipMemcpyDeviceToDevice);
    hipMemcpy(dev_gridIndicesCopy2, dev_particleGridIndices, numObjects * sizeof(int), hipMemcpyDeviceToDevice);

    // Wrap device vectors in thrust iterators for use with thrust.
    dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);
    dev_thrust_gridIndicesCopy1 = thrust::device_ptr<int>(dev_gridIndicesCopy1);
    dev_thrust_gridIndicesCopy2 = thrust::device_ptr<int>(dev_gridIndicesCopy2);
    dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);
    dev_thrust_posArrayIndices = thrust::device_ptr<glm::vec3>(dev_pos);
    dev_thrust_velArrayIndices = thrust::device_ptr<glm::vec3>(dev_vel1);
    //thrust::sort_by_key including velocity and position
    thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);
    thrust::sort_by_key(dev_thrust_gridIndicesCopy1, dev_thrust_gridIndicesCopy1 + numObjects, dev_thrust_posArrayIndices);
    thrust::sort_by_key(dev_thrust_gridIndicesCopy2, dev_thrust_gridIndicesCopy2 + numObjects, dev_thrust_velArrayIndices);

    kernResetIntBuffer << <fullBlocksForCell, blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1);
    checkCUDAErrorWithLine("kernResetIntBuffer failed!");

    kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
    checkCUDAErrorWithLine("kernIdentifyCellStartEnd failed!");

    kernUpdateVelNeighborSearchCoherent << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum,
        gridInverseCellWidth, gridCellWidth, gridCellCount, dev_gridCellStartIndices, dev_gridCellEndIndices,
        dev_pos, dev_vel1, dev_vel2);
    checkCUDAErrorWithLine("kernUpdateVelNeighborSearchScattered failed!");

    kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt, dev_pos, dev_vel2);
    checkCUDAErrorWithLine("kernUpdatePos failed!");

    glm::vec3* temp = dev_vel1;
    dev_vel1 = dev_vel2;
    dev_vel2 = temp;
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);
  hipFree(dev_gridIndicesCopy1);
  hipFree(dev_gridIndicesCopy2);
  checkCUDAErrorWithLine("hipFree failed!");
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
