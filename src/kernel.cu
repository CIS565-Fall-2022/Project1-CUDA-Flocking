#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"
#include <>

// potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.

// Grid parameters based on simulation parameters.
// These are automatically computed in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* basic helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");
  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");
  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects, dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, numObjects * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");
  hipMalloc((void**)&dev_particleGridIndices, numObjects * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");
  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");
  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);
  dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3* pos, float* vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3* vel, float* vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float* vbodptr_positions, float* vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3* pos, const glm::vec3* vel) {
  // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
  // Rule 2: boids try to stay a distance d away from each other
  // Rule 3: boids try to match the speed of surrounding boids
  glm::vec3 perceivedCenter = glm::vec3(0.f);
  glm::vec3 c = glm::vec3(0.f);
  glm::vec3 perceivedVel = glm::vec3(0.f);
  int neighborInRule1Dist = 0;
  int neighborInRule3Dist = 0;
  for (int i = 0; i < N; ++i) {
    if (i != iSelf) {
      float dist = glm::distance(pos[i], pos[iSelf]);
      if (dist < rule1Distance) {
        perceivedCenter += pos[i];
        ++neighborInRule1Dist;
      }
      if (dist < rule2Distance) {
        c -= (pos[i] - pos[iSelf]);
      }
      if (dist < rule3Distance) {
        perceivedVel += vel[i];
        ++neighborInRule3Dist;
      }
    }
  }

  glm::vec3 velChange = glm::vec3(0.f);
  if (neighborInRule1Dist > 0) {
    perceivedCenter /= neighborInRule1Dist;
    velChange += (perceivedCenter - pos[iSelf]) * rule1Scale;
  }
  velChange += c * rule2Scale;
  if (neighborInRule3Dist > 0) {
    perceivedVel /= neighborInRule3Dist;
    velChange += perceivedVel * rule3Scale;
  }
  return velChange;
}

/**
* implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (i >= N) { return; }
  // Compute a new velocity based on pos and vel1
  // Record in new vel2. (since vel1 may still need to be referenced for rule3)
  vel2[i] = vel1[i] + computeVelocityChange(N, i, pos, vel1);

  // Clamp speed if needed
  if (glm::length(vel2[i]) > maxSpeed) {
    vel2[i] = glm::normalize(vel2[i]) * maxSpeed;
  }
}

/**
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, glm::vec3* pos, int* indices, int* gridIndices) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (i >= N) { return; }
  // parallel array of indices as pointers to the actual boid data in pos and vel1/vel2
  indices[i] = i;
  // Label each boid with the index of its grid cell.
  glm::vec3 posInCell = (pos[i] - gridMin) * inverseCellWidth;
  gridIndices[i] = gridIndex3Dto1D(int(posInCell.x), int(posInCell.y), int(posInCell.z),
    gridResolution);
}

// indicating that a cell does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (i >= N) { return; }
  // Identify the start point of each cell in the gridIndices array.
  // parallel unrolling of a loop
  int currBoidIdx = particleGridIndices[i]; // index IN dev_pos or whatever
  if (i == N - 1) {
    gridCellEndIndices[currBoidIdx] = i;
    return;
  }

  int nextBoidIdx = particleGridIndices[i + 1];
  if (i == 0) {
    gridCellStartIndices[currBoidIdx] = i;
  }
  // "this index doesn't match the one before it, must be a new cell!"
  if (currBoidIdx != nextBoidIdx) {
    gridCellEndIndices[currBoidIdx] = i + 1;
    gridCellStartIndices[nextBoidIdx] = i + 1;
  } 
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // Update a boid's velocity using the uniform grid
  int currIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (currIdx >= N) { return; }
  // - Identify the grid cell that this particle is in
  glm::vec3 posRelativeToGrid = pos[currIdx] - gridMin;
  glm::vec3 posInCell = posRelativeToGrid * inverseCellWidth;
  glm::vec3 posOfCell = glm::vec3(int(posInCell.x), int(posInCell.y), int(posInCell.z));
  int gridCellIdx = gridIndex3Dto1D(posOfCell.x, posOfCell.y, posOfCell.z, gridResolution);
  // Identify which cells may contain neighbors. This isn't always 8.
  glm::vec3 posRelativeToCell = posRelativeToGrid - (posOfCell * cellWidth);
  int xSearchDir = posRelativeToCell.x > cellWidth / 2 ? 1 : -1;
  int ySearchDir = posRelativeToCell.y > cellWidth / 2 ? 1 : -1;
  int zSearchDir = posRelativeToCell.z > cellWidth / 2 ? 1 : -1;

  //For each cell, read the start/end indices in the boid pointer array.
  glm::vec3 perceivedCenter = glm::vec3(0.f);
  glm::vec3 c = glm::vec3(0.f);
  glm::vec3 perceivedVel = glm::vec3(0.f);
  int neighborInRule1Dist = 0;
  int neighborInRule3Dist = 0;
  for (int x = 0; abs(x) <= abs(xSearchDir); ++x) {
    for (int y = 0; abs(y) <= abs(ySearchDir); ++y) {
      for (int z = 0; abs(z) <= abs(zSearchDir); ++z) {
        int neighborCellIdx = gridCellIdx + 
          x + y * gridResolution + z * gridResolution * gridResolution;
        // gridRes ^ 3 == gridCellCount
        if (neighborCellIdx < 0 || 
          gridResolution * gridResolution * gridResolution <= neighborCellIdx) { continue; }
        int gridStart = gridCellStartIndices[neighborCellIdx];
        if (gridStart == -1) { continue; }
        // Access each boid in the cell, compute vel change from boids rules
        for (int arrIdx = gridStart; arrIdx < gridCellEndIndices[neighborCellIdx]; ++arrIdx) {
          int neighborIdx = particleArrayIndices[arrIdx];
          if (currIdx == neighborIdx) { continue; }
          // Rule 1: boids fly towards their local perceived center of mass
          // Rule 2: boids try to stay a distance d away from each other
          // Rule 3: boids try to match the speed of surrounding boids
          float dist = glm::distance(pos[neighborIdx], pos[currIdx]);
          if (dist < rule1Distance) {
            perceivedCenter += pos[neighborIdx];
            ++neighborInRule1Dist;
          }
          if (dist < rule2Distance) {
            c -= (pos[neighborIdx] - pos[currIdx]);
          }
          if (dist < rule3Distance) {
            perceivedVel += vel1[neighborIdx];
            ++neighborInRule3Dist;
          }
        }
      }
    }
  }
  glm::vec3 velChange = glm::vec3(0.f);
  if (neighborInRule1Dist > 0) {
    perceivedCenter /= neighborInRule1Dist;
    velChange += (perceivedCenter - pos[currIdx]) * rule1Scale;
  }
  velChange += c * rule2Scale;
  if (neighborInRule3Dist > 0) {
    perceivedVel /= neighborInRule3Dist;
    velChange += perceivedVel * rule3Scale;
  }
  vel2[currIdx] = vel1[currIdx] + velChange;
  // Clamp speed if needed
  if (glm::length(vel2[currIdx]) > maxSpeed) {
    vel2[currIdx] = glm::normalize(vel2[currIdx]) * maxSpeed;
  }
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // use kernels to step the simulation forward in time.
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
  kernUpdateVelocityBruteForce<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, dev_vel1, dev_vel2);
  checkCUDAErrorWithLine("kernUpdateVelocityBruteForce failed!");
  // ping-pong velocity buffers
  std::swap(dev_vel1, dev_vel2);
  kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel1);
  checkCUDAErrorWithLine("kernUpdatePos failed!");
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // Uniform Grid Neighbor search using Thrust sort.
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids. (should have already been done in init ?)
  kernComputeIndices<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount,
    gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices,
    dev_particleGridIndices);
  checkCUDAErrorWithLine("kernComputeIndices failed!");
  // Unstable key sort using Thrust. stable sort isn't necessary
  thrust::sort_by_key(dev_thrust_particleGridIndices,
    dev_thrust_particleGridIndices + numObjects,
    dev_thrust_particleArrayIndices);

  // Naively unroll the loop for finding the start and end indices of each
  // cell's data pointers in the array of boid indices
  dim3 fullBlocksPerCell((gridCellCount + blockSize - 1) / blockSize);
  kernResetIntBuffer<<<fullBlocksPerCell, blockSize>>>(gridCellCount,
    dev_gridCellStartIndices, -1);
  checkCUDAErrorWithLine("kernResetIntBuffer failed!");
  kernIdentifyCellStartEnd<<<fullBlocksPerGrid, blockSize>>>(numObjects,
    dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
  checkCUDAErrorWithLine("kernIdentifyCellStartEnd failed!");
  // velocity updates using neighbor search
  kernUpdateVelNeighborSearchScattered<<<fullBlocksPerGrid, blockSize>>>(numObjects,
    gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth,
    dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices,
    dev_pos, dev_vel1, dev_vel2);
  checkCUDAErrorWithLine("kernUpdateVelNeighborSearchScattered failed!");
  // ping poing, Update positions
  std::swap(dev_vel1, dev_vel2);
  kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel1);
  checkCUDAErrorWithLine("kernUpdatePos failed!");
}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);
}

void Boids::unitTest() {
  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
