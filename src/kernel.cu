#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 2.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// 
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell? first and last index 

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
glm::vec3 *dev_coherent_pos;
glm::vec3 *dev_coherent_vel1;
glm::vec3* dev_coherent_vel2;
// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;//easier to do multiply than divide
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");
  hipMalloc((void**)&dev_particleGridIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  hipMalloc((void**)&dev_gridCellStartIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");
  hipMalloc((void**)&dev_gridCellEndIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  hipMalloc((void**)&dev_coherent_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_coherent_pos failed!");
  hipMalloc((void**)&dev_coherent_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_coherent_vel1 failed!");
  hipMalloc((void**)&dev_coherent_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_coherent_vel2 failed!");

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
    if (iSelf >= N) {
        return;
    }
    glm::vec3 newVel1,newVel2, center = glm::vec3(0.0f, 0.0f, 0.0f);
    int neightborCount1 = 0;
    int neightborCount2 = 0;
    for (int i = 0; i < N; i++) {
        if (i == iSelf) continue;
        float distance = glm::distance(pos[iSelf], pos[i]);
        // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
        if (distance < rule1Distance) {
            neightborCount1++;
            center += pos[i];
        }
        // Rule 2: boids try to stay a distance d away from each other
        if (distance < rule2Distance) {
            newVel1 -= (pos[i] - pos[iSelf]);
        }
        // Rule 3: boids try to match the speed of surrounding boids
        if (distance < rule3Distance) {
            neightborCount2++;
            newVel2 += vel[i];
        }
    }
    if (neightborCount1 != 0) {
      center /= neightborCount1;
      center -= pos[iSelf];
      center *= rule1Scale;
    }
    if (neightborCount2 != 0) {
      newVel2 /= neightborCount2;
      newVel2 *= rule3Scale;
    }
    newVel1 *= rule2Scale;
    return (center + newVel1 + newVel2);
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) {
        return;
    }
  // Compute a new velocity based on pos and vel1
       glm::vec3 velocity = vel1[index] + computeVelocityChange(N, index, pos, vel1);
  // Clamp the speed
       float speedMag = glm::length(velocity);
       float speedMagRatio = glm::clamp(speedMag, 0.0f, maxSpeed) / speedMag;
  // Record the new velocity into vel2. Question: why NOT vel1? -> Because we are using ping-pong buffer and while we read from vel1, the ouput channel should be vel2;
  // and we will swap these buffers later
       vel2[index] = velocity * speedMagRatio;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;
  float x = thisPos.x;
  float y = thisPos.y;
  float z = thisPos.z;
  float x1 = vel[index].x;
  float y1 = vel[index].y;
  float z1 = vel[index].z; 
  pos[index] = thisPos;

}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
//for (z) for (y) for (x) so the memory travelling will be shorter
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

//overload this to get the 1D index directly from pos
__device__ int gridIndex3Dto1D(float x, float y, float z, glm::vec3 gridMin, int gridResolution, float gridInverseCellWidth) {
    return gridIndex3Dto1D(floor((x-gridMin.x) * gridInverseCellWidth), floor((y-gridMin.y) * gridInverseCellWidth), floor((z-gridMin.z) * gridInverseCellWidth), gridResolution);
}


//edge detection for 2.1
__device__ bool edgeDetection(float x, float y, float z, glm::vec3 gridMin, int gridResolution, float gridCellWidth) {
    x -= gridMin.x; y -= gridMin.y; z -= gridMin.z;
    if (x < 0 || x > gridResolution*gridCellWidth) return false;
    if (y < 0 || y > gridResolution*gridCellWidth) return false;
    if (z < 0 || z > gridResolution*gridCellWidth) return false;
    return true;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) {
        return;
    }
    glm::vec3 distance = pos[index] - gridMin;
    int gridIndex = gridIndex3Dto1D(floor(distance.x*inverseCellWidth), floor(distance.y*inverseCellWidth), floor(distance.z*inverseCellWidth), gridResolution);
    gridIndices[index] = gridIndex;
    indices[index] = index;
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) {
        return;
    }
    if (!index || particleGridIndices[index - 1] != particleGridIndices[index]) {
        gridCellStartIndices[particleGridIndices[index]] = index;
    }
    if (index == N-1 || particleGridIndices[index+1] != particleGridIndices[index]) {
        gridCellEndIndices[particleGridIndices[index]] = index + 1;
    }
}

__global__ void kernUpdateVelNeighborSearchScattered(
    int N, int gridResolution, glm::vec3 gridMin,
    float inverseCellWidth, float cellWidth,
    int* gridCellStartIndices, int* gridCellEndIndices,
    int* particleArrayIndices,
    glm::vec3* pos, glm::vec3* vel1, glm::vec3* vel2) {
    // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
    // the number of boids that need to be checked.
    // - Identify the grid cell that this particle is in
    // - Identify which cells may contain neighbors. This isn't always 8.
    // - For each cell, read the start/end indices in the boid pointer array.
    // - Access each boid in the cell and compute velocity change from
    //   the boids rules, if this boid is within the neighborhood distance.
    // - Clamp the speed change before putting the new speed in vel2

    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) {
        return;
    }
    glm::vec3 newVel1, newVel2, center = glm::vec3(0.0f, 0.0f, 0.0f);
    int neightborCount1 = 0;
    int neightborCount2 = 0;
    //For this I am trying to use grid-looping since this is the most efficient way to solve(at least thats what I can think of)
    //first we find the n grids using for loop
    //then we go over the gridIdCollection to get all the boids
    float maxDistance = imax(rule1Distance, imax(rule2Distance, rule3Distance));
    for (float posZ = pos[index].z - maxDistance; posZ <= pos[index].z + maxDistance; posZ += cellWidth) {
        for (float posY = pos[index].y - maxDistance; posY <= pos[index].y + maxDistance; posY += cellWidth) {
            for (float posX = pos[index].x - maxDistance; posX <= pos[index].x + maxDistance; posX += cellWidth) {
                //first we did edge detection
                if (edgeDetection(posX, posY, posZ, gridMin, gridResolution, cellWidth)) {
                    int gridId = gridIndex3Dto1D(posX, posY, posZ, gridMin, gridResolution, inverseCellWidth);
                    if (gridId >= 0 && gridId < gridResolution * gridResolution * gridResolution) {
                        int start = gridCellStartIndices[gridId];
                        int end = gridCellEndIndices[gridId];
                        for (int i = start; i < end; i++) {
                            int particleNum = particleArrayIndices[i];
                            if (i != index) {
                                float distance = glm::distance(pos[index], pos[particleNum]);
                                // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
                                if (distance < rule1Distance) {
                                    neightborCount1++;
                                    center += pos[particleNum];
                                }
                                // Rule 2: boids try to stay a distance d away from each other
                                if (distance < rule2Distance) {
                                    newVel1 -= (pos[particleNum] - pos[index]);
                                }
                                // Rule 3: boids try to match the speed of surrounding boids
                                if (distance < rule3Distance) {
                                    neightborCount2++;
                                    newVel2 += vel1[particleNum];
                                }
                            }
                        }
                    }
                }
            }
        }
    }
    if (neightborCount1 != 0) {
        center /= neightborCount1;
        center -= pos[index];
        center *= rule1Scale;
    }
    if (neightborCount2 != 0) {
        newVel2 /= neightborCount2;
        newVel2 *= rule3Scale;
    }
    newVel1 *= rule2Scale;
    //clamp and mark
    glm::vec3 velocity = vel1[index] + center + newVel1 + newVel2;
    float speedMag = glm::length(velocity);
    float speedMagRatio = glm::clamp(speedMag, 0.0f, maxSpeed) / speedMag;
    vel2[index] = velocity * speedMagRatio;
}

__global__ void kernSetCoherentBuffers(int N, int* particleArrayIndices, glm::vec3* coherent_posIndices, glm::vec3* coherent_velIndices, glm::vec3* pos, glm::vec3* vel) {
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) {
        return;
    }
    coherent_posIndices[index] = pos[particleArrayIndices[index]];
    coherent_velIndices[index] = vel[particleArrayIndices[index]];
}

__global__ void kernUnsetCoherentBuffers(int N, int* particleArrayIndices, glm::vec3* coherent_posIndices, glm::vec3* coherent_velIndices, glm::vec3* pos, glm::vec3* vel) {
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) {
        return;
    }
    pos[particleArrayIndices[index]] = coherent_posIndices[index];
    vel[particleArrayIndices[index]] = coherent_velIndices[index];
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
  // - Clamp the speed change before putting the new speed in vel2
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) {
        return;
    }
    glm::vec3 newVel1, newVel2, center = glm::vec3(0.0f, 0.0f, 0.0f);
    int neightborCount1 = 0;
    int neightborCount2 = 0;

    float maxDistance = imax(rule1Distance, imax(rule2Distance, rule3Distance));
    for (float posZ = pos[index].z - maxDistance; posZ <= pos[index].z + maxDistance; posZ += cellWidth) {
        for (float posY = pos[index].y - maxDistance; posY <= pos[index].y + maxDistance; posY += cellWidth) {
            for (float posX = pos[index].x - maxDistance; posX <= pos[index].x + maxDistance; posX += cellWidth) {
                //first we did edge detection
                if (edgeDetection(posX, posY, posZ, gridMin, gridResolution, cellWidth)) {
                    int gridId = gridIndex3Dto1D(posX, posY, posZ, gridMin, gridResolution, inverseCellWidth);
                    if (gridId >= 0 && gridId < gridResolution * gridResolution * gridResolution) {
                        int start = gridCellStartIndices[gridId];
                        int end = gridCellEndIndices[gridId];
                        for (int i = start; i < end; i++) {
                            if (i != index) {
                                float distance = glm::distance(pos[index], pos[i]);
                                // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
                                if (distance < rule1Distance) {
                                    neightborCount1++;
                                    center += pos[i];
                                }
                                // Rule 2: boids try to stay a distance d away from each other
                                if (distance < rule2Distance) {
                                    newVel1 -= (pos[i] - pos[index]);
                                }
                                // Rule 3: boids try to match the speed of surrounding boids
                                if (distance < rule3Distance) {
                                    neightborCount2++;
                                    newVel2 += vel1[i];
                                }
                            }
                        }
                    }
                }
            }
        }
    }
    if (neightborCount1 != 0) {
        center /= neightborCount1;
        center -= pos[index];
        center *= rule1Scale;
    }
    if (neightborCount2 != 0) {
        newVel2 /= neightborCount2;
        newVel2 *= rule3Scale;
    }
    newVel1 *= rule2Scale;
    //clamp and mark
    glm::vec3 velocity = vel1[index] + center + newVel1 + newVel2;
    float speedMag = glm::length(velocity);
    float speedMagRatio = glm::clamp(speedMag, 0.0f, maxSpeed) / speedMag;
    vel2[index] = velocity * speedMagRatio;
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
    kernUpdateVelocityBruteForce << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_pos, dev_vel1, dev_vel2);
    checkCUDAErrorWithLine("kernUpdateVelocityBruteForces failed!");
    kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt, dev_pos, dev_vel2);
    checkCUDAErrorWithLine("kernUpdatePos failed!");
  // TODO-1.2 ping-pong the velocity buffers
    /*glm::vec3 * velReplace = dev_vel2;
    dev_vel2 = dev_vel1;
    dev_vel1 = velReplace;*/
    std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationScatteredGrid(float dt) {
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - Perform velocity updates using neighbor search
  // - Update positions
  // - Ping-pong buffers as needed
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
    kernComputeIndices << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
    //thrust sort
    // Wrap device vectors in thrust iterators for use with thrust.
    dev_thrust_particleArrayIndices = thrust::device_ptr<int>{ dev_particleArrayIndices };
    dev_thrust_particleGridIndices = thrust::device_ptr<int>{ dev_particleGridIndices };
    // LOOK-2.1 Example for using thrust::sort_by_key
    thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

    dim3 fullGrid((gridCellCount + blockSize - 1) / blockSize);
    kernResetIntBuffer << <fullGrid, blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1);
    kernResetIntBuffer << <fullGrid, blockSize >> > (gridCellCount, dev_gridCellEndIndices, -1);

    kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

    kernUpdateVelNeighborSearchScattered << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);
    kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt, dev_pos, dev_vel2);

    //pingpong
    std::swap(dev_vel1, dev_vel2);

}

void Boids::stepSimulationCoherentGrid(float dt) {
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
  // - Perform velocity updates using neighbor search
  // - Update positions
  
    dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
    kernComputeIndices << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
    //thrust sort
    // Wrap device vectors in thrust iterators for use with thrust.
    dev_thrust_particleArrayIndices = thrust::device_ptr<int>{ dev_particleArrayIndices };
    dev_thrust_particleGridIndices = thrust::device_ptr<int>{ dev_particleGridIndices };
    // LOOK-2.1 Example for using thrust::sort_by_key
    thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

    dim3 fullGrid((gridCellCount + blockSize - 1) / blockSize);
    kernResetIntBuffer << <fullGrid, blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1);
    kernResetIntBuffer << <fullGrid, blockSize >> > (gridCellCount, dev_gridCellEndIndices, -1);

    kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

    //after this we start fill coherent buffers; for 2.3 we do not need to change between vel1 and vel2, instead we will first get dev_vel1 into dev_coherent_vel1Indices, then output as coherent_vel2Indices, and then pingpong to dev_vel1;
    kernSetCoherentBuffers << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleArrayIndices, dev_coherent_pos, dev_coherent_vel1, dev_pos, dev_vel1);

    kernUpdateVelNeighborSearchCoherent << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_coherent_pos, dev_coherent_vel1, dev_coherent_vel2);
    kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt, dev_coherent_pos, dev_coherent_vel2);

    // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
    kernUnsetCoherentBuffers << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleArrayIndices, dev_coherent_pos, dev_coherent_vel2, dev_pos, dev_vel1);
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);
  hipFree(dev_coherent_pos);
  hipFree(dev_coherent_vel1);
  hipFree(dev_coherent_vel2);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
